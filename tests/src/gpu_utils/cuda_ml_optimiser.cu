#include "hip/hip_runtime.h"
#include <sys/time.h>
#include <stdio.h>
#include <time.h>
#include <math.h>
#include <ctime>
#include <vector>
#include <iostream>
#include "src/gpu_utils/cuda_projector.h"
#include "src/gpu_utils/cuda_projector.cuh"
#include "src/gpu_utils/cuda_projector_plan.h"
#include "src/gpu_utils/cuda_benchmark_utils.h"
#include "src/gpu_utils/cuda_ml_optimiser.h"
#include "src/gpu_utils/cuda_kernels/helper.cuh"
#include "src/gpu_utils/cuda_kernels/diff2.cuh"
#include "src/gpu_utils/cuda_kernels/wavg.cuh"
#include "src/gpu_utils/cuda_helper_functions.cuh"
#include "src/gpu_utils/cuda_mem_utils.h"
#include "src/complex.h"
#include "src/helix.h"
#include "src/error.h"
#include <fstream>
#include <hip/hip_runtime.h>
#include "src/parallel.h"
#include <signal.h>
#include <map>

#ifdef CUDA_FORCESTL
#include "src/gpu_utils/cuda_utils_stl.cuh"
#else
#include "src/gpu_utils/cuda_utils_cub.cuh"
#endif

static pthread_mutex_t global_mutex = PTHREAD_MUTEX_INITIALIZER;

void getFourierTransformsAndCtfs(long int my_ori_particle,
		OptimisationParamters &op,
		SamplingParameters &sp,
		MlOptimiser *baseMLO,
		MlOptimiserCuda *cudaMLO
		)
{
		GTIC(cudaMLO->timer,"getFourierTransformsAndCtfs");
		//cudaMLO->timer.cuda_gpu_tic("getFourierTransformsAndCtfs");
#ifdef TIMING
	if (op.my_ori_particle == baseMLO->exp_my_first_ori_particle)
		baseMLO->timer.tic(baseMLO->TIMING_ESP_FT);
#endif
	//FourierTransformer transformer;
	CUSTOM_ALLOCATOR_REGION_NAME("GFTCTF");

	for (int ipart = 0; ipart < baseMLO->mydata.ori_particles[my_ori_particle].particles_id.size(); ipart++)
	{
		CTIC(cudaMLO->timer,"init");
		FileName fn_img;
		Image<RFLOAT> img, rec_img;
		MultidimArray<Complex > Fimg;
		MultidimArray<Complex > Faux(cudaMLO->transformer.fFourier,true);
		MultidimArray<RFLOAT> Fctf;

		// What is my particle_id?
		long int part_id = baseMLO->mydata.ori_particles[my_ori_particle].particles_id[ipart];
		// Which group do I belong?
		int group_id =baseMLO->mydata.getGroupId(part_id);

		// Get the right line in the exp_fn_img strings (also exp_fn_recimg and exp_fn_ctfs)
		int istop = 0;
		for (long int ii = baseMLO->exp_my_first_ori_particle; ii < my_ori_particle; ii++)
			istop += baseMLO->mydata.ori_particles[ii].particles_id.size();
		istop += ipart;

		if (!baseMLO->mydata.getImageNameOnScratch(part_id, fn_img))
		{
			std::istringstream split(baseMLO->exp_fn_img);
			for (int i = 0; i <= istop; i++)
				getline(split, fn_img);
		}
		sp.current_img = fn_img;

		// Get the norm_correction
		RFLOAT normcorr = DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_NORM);

		// Get the optimal origin offsets from the previous iteration
		Matrix1D<RFLOAT> my_old_offset(2), my_prior(2);
		XX(my_old_offset) = DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_XOFF);
		YY(my_old_offset) = DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_YOFF);
		XX(my_prior)      = DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_XOFF_PRIOR);
		YY(my_prior)      = DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_YOFF_PRIOR);
		// Uninitialised priors were set to 999.
		if (XX(my_prior) > 998.99 && XX(my_prior) < 999.01)
			XX(my_prior) = 0.;
		if (YY(my_prior) > 998.99 && YY(my_prior) < 999.01)
			YY(my_prior) = 0.;

		if (cudaMLO->dataIs3D)
		{
			my_old_offset.resize(3);
			my_prior.resize(3);
			ZZ(my_old_offset) = DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_ZOFF);
			ZZ(my_prior)      = DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_ZOFF_PRIOR);
			// Unitialised priors were set to 999.
			if (ZZ(my_prior) > 998.99 && ZZ(my_prior) < 999.01)
				ZZ(my_prior) = 0.;
		}
		CTOC(cudaMLO->timer,"init");

		CTIC(cudaMLO->timer,"nonZeroProb");
		if (baseMLO->mymodel.orientational_prior_mode != NOPRIOR && !(baseMLO->do_skip_align ||baseMLO-> do_skip_rotate))
		{
			// First try if there are some fixed prior angles
			RFLOAT prior_rot = DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_ROT_PRIOR);
			RFLOAT prior_tilt = DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_TILT_PRIOR);
			RFLOAT prior_psi = DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_PSI_PRIOR);
			RFLOAT prior_psi_flip_ratio =  (baseMLO->mymodel.nr_bodies > 1 ) ? 0. : DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_PSI_PRIOR_FLIP_RATIO);

			bool do_auto_refine_local_searches = (baseMLO->do_auto_refine) && (baseMLO->sampling.healpix_order >= baseMLO->autosampling_hporder_local_searches);
			bool do_classification_local_searches = (! baseMLO->do_auto_refine) && (baseMLO->mymodel.orientational_prior_mode == PRIOR_ROTTILT_PSI)
					&& (baseMLO->mymodel.sigma2_rot > 0.) && (baseMLO->mymodel.sigma2_tilt > 0.) && (baseMLO->mymodel.sigma2_psi > 0.);
			bool do_local_angular_searches = (do_auto_refine_local_searches) || (do_classification_local_searches);

			// If there were no defined priors (i.e. their values were 999.), then use the "normal" angles
			if (prior_rot > 998.99 && prior_rot < 999.01)
				prior_rot = DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_ROT);
			if (prior_tilt > 998.99 && prior_tilt < 999.01)
				prior_tilt = DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_TILT);
			if ( (baseMLO->do_helical_refine) && (baseMLO->helical_keep_tilt_prior_fixed) && (do_local_angular_searches) )
				prior_tilt = DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_TILT);
			if (prior_psi > 998.99 && prior_psi < 999.01)
				prior_psi = DIRECT_A2D_ELEM(baseMLO->exp_metadata,op. metadata_offset + ipart, METADATA_PSI);
			if (prior_psi_flip_ratio > 998.99 && prior_psi_flip_ratio < 999.01)
				prior_psi_flip_ratio = 0.5;

			////////// How does this work now: each particle has a different sampling object?!!!
			// Select only those orientations that have non-zero prior probability

			if (baseMLO->do_helical_refine)
			{
				baseMLO->sampling.selectOrientationsWithNonZeroPriorProbabilityFor3DHelicalReconstruction(prior_rot, prior_tilt, prior_psi,
										sqrt(baseMLO->mymodel.sigma2_rot), sqrt(baseMLO->mymodel.sigma2_tilt), sqrt(baseMLO->mymodel.sigma2_psi),
										op.pointer_dir_nonzeroprior, op.directions_prior, op.pointer_psi_nonzeroprior, op.psi_prior,
										do_local_angular_searches, prior_psi_flip_ratio);
			}
			else
			{
				baseMLO->sampling.selectOrientationsWithNonZeroPriorProbability(prior_rot, prior_tilt, prior_psi,
						sqrt(baseMLO->mymodel.sigma2_rot), sqrt(baseMLO->mymodel.sigma2_tilt), sqrt(baseMLO->mymodel.sigma2_psi),
						op.pointer_dir_nonzeroprior, op.directions_prior, op.pointer_psi_nonzeroprior, op.psi_prior);
			}

			long int nr_orients = baseMLO->sampling.NrDirections(0, &op.pointer_dir_nonzeroprior) * baseMLO->sampling.NrPsiSamplings(0, &op.pointer_psi_nonzeroprior);
			if (nr_orients == 0)
			{
				std::cerr << " sampling.NrDirections()= " << baseMLO->sampling.NrDirections(0, &op.pointer_dir_nonzeroprior)
						<< " sampling.NrPsiSamplings()= " << baseMLO->sampling.NrPsiSamplings(0, &op.pointer_psi_nonzeroprior) << std::endl;
				REPORT_ERROR("Zero orientations fall within the local angular search. Increase the sigma-value(s) on the orientations!");
			}

		}
		CTOC(cudaMLO->timer,"nonZeroProb");

		CTIC(cudaMLO->timer,"setXmippOrigin1");
		// Get the image and recimg data
		if (baseMLO->do_parallel_disc_io)
		{

			// If all slaves had preread images into RAM: get those now
			if (baseMLO->do_preread_images)
			{

                img().reshape(baseMLO->mydata.particles[part_id].img);
                CTIC(cudaMLO->timer,"ParaReadPrereadImages");
				FOR_ALL_DIRECT_ELEMENTS_IN_MULTIDIMARRAY(baseMLO->mydata.particles[part_id].img)
				{
                	DIRECT_MULTIDIM_ELEM(img(), n) = (RFLOAT)DIRECT_MULTIDIM_ELEM(baseMLO->mydata.particles[part_id].img, n);
				}
				CTOC(cudaMLO->timer,"ParaReadPrereadImages");
			}
			else
			{
				if (cudaMLO->dataIs3D)
				{
					CTIC(cudaMLO->timer,"ParaRead3DImages");
					img.read(fn_img);
					img().setXmippOrigin();
					CTOC(cudaMLO->timer,"ParaRead3DImages");
				}
				else
				{
					CTIC(cudaMLO->timer,"ParaRead2DImages");
					img() = baseMLO->exp_imgs[istop];
					CTOC(cudaMLO->timer,"ParaRead2DImages");
				}
			}
			if (baseMLO->has_converged && baseMLO->do_use_reconstruct_images)
			{
				FileName fn_recimg;
				std::istringstream split2(baseMLO->exp_fn_recimg);
				// Get the right line in the exp_fn_img string
				for (int i = 0; i <= istop; i++)
					getline(split2, fn_recimg);
				rec_img.read(fn_recimg);
				rec_img().setXmippOrigin();
			}
		}
		else
		{
			// Unpack the image from the imagedata
			if (cudaMLO->dataIs3D)
			{
				CTIC(cudaMLO->timer,"Read3DImages");
				CTIC(cudaMLO->timer,"resize");
				img().resize(baseMLO->mymodel.ori_size, baseMLO->mymodel.ori_size,baseMLO-> mymodel.ori_size);
				CTOC(cudaMLO->timer,"resize");
				// Only allow a single image per call of this function!!! nr_pool needs to be set to 1!!!!
				// This will save memory, as we'll need to store all translated images in memory....
				FOR_ALL_DIRECT_ELEMENTS_IN_ARRAY3D(img())
				{
					DIRECT_A3D_ELEM(img(), k, i, j) = DIRECT_A3D_ELEM(baseMLO->exp_imagedata, k, i, j);
				}
				img().setXmippOrigin();

				if (baseMLO->has_converged && baseMLO->do_use_reconstruct_images)
				{
					rec_img().resize(baseMLO->mymodel.ori_size, baseMLO->mymodel.ori_size,baseMLO-> mymodel.ori_size);
					int offset = (baseMLO->do_ctf_correction) ? 2 * baseMLO->mymodel.ori_size : baseMLO->mymodel.ori_size;
					FOR_ALL_DIRECT_ELEMENTS_IN_ARRAY3D(rec_img())
					{
						DIRECT_A3D_ELEM(rec_img(), k, i, j) = DIRECT_A3D_ELEM(baseMLO->exp_imagedata, offset + k, i, j);
					}
					rec_img().setXmippOrigin();

				}
				CTOC(cudaMLO->timer,"Read3DImages");

			}
			else
			{
				CTIC(cudaMLO->timer,"Read2DImages");
				img().resize(baseMLO->mymodel.ori_size, baseMLO->mymodel.ori_size);
				FOR_ALL_DIRECT_ELEMENTS_IN_ARRAY2D(img())
				{
					DIRECT_A2D_ELEM(img(), i, j) = DIRECT_A3D_ELEM(baseMLO->exp_imagedata, op.metadata_offset + ipart, i, j);
				}
				img().setXmippOrigin();
				if (baseMLO->has_converged && baseMLO->do_use_reconstruct_images)
				{

					////////////// TODO: think this through for no-threads here.....
					rec_img().resize(baseMLO->mymodel.ori_size, baseMLO->mymodel.ori_size);
					FOR_ALL_DIRECT_ELEMENTS_IN_ARRAY2D(rec_img())
					{
						DIRECT_A2D_ELEM(rec_img(), i, j) = DIRECT_A3D_ELEM(baseMLO->exp_imagedata, baseMLO->exp_nr_images + op.metadata_offset + ipart, i, j);
					}
					rec_img().setXmippOrigin();
				}
				CTOC(cudaMLO->timer,"Read2DImages");
			}
		}
		CTOC(cudaMLO->timer,"setXmippOrigin1");

		CTIC(cudaMLO->timer,"selfTranslate");

		/* FIXME :  For some reason the device-allocation inside "selfTranslate" takes a much longer time than expected.
		 * 			I tried moving it up and placing the size under a bunch of if()-cases, but this simply transferred the
		 * 			allocation-cost to that region. /BjoernF,160129
		 */

		// Apply (rounded) old offsets first
		my_old_offset.selfROUND();

		int img_size = img.data.nzyxdim;
		CudaGlobalPtr<XFLOAT> d_img(img_size,0,cudaMLO->devBundle->allocator);
		CudaGlobalPtr<XFLOAT> temp(img_size,0,cudaMLO->devBundle->allocator);
		d_img.device_alloc();
		temp.device_alloc();
		d_img.device_init(0);

		for (int i=0; i<img_size; i++)
			temp[i] = img.data.data[i];

		temp.cp_to_device();
		temp.streamSync();

		int STBsize = ( (int) ceilf(( float)img_size /(float)BLOCK_SIZE));
		// Apply the norm_correction term
		if (baseMLO->do_norm_correction)
		{
			CTIC(cudaMLO->timer,"norm_corr");
			cuda_kernel_multi<<<STBsize,BLOCK_SIZE>>>(
									~temp,
									(XFLOAT)(baseMLO->mymodel.avg_norm_correction / normcorr),
									img_size);
			LAUNCH_PRIVATE_ERROR(hipGetLastError(),cudaMLO->errorStatus);
			temp.streamSync();
			CTOC(cudaMLO->timer,"norm_corr");
		}


		// Helical reconstruction: calculate old_offset in the system of coordinates of the helix, i.e. parallel & perpendicular, depending on psi-angle!
		// For helices do NOT apply old_offset along the direction of the helix!!
		Matrix1D<RFLOAT> my_old_offset_helix_coords;
		RFLOAT rot_deg = DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_ROT);
		RFLOAT tilt_deg = DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_TILT);
		RFLOAT psi_deg = DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_PSI);
		if ( (baseMLO->do_helical_refine) && (! baseMLO->ignore_helical_symmetry) )
		{
			// Calculate my_old_offset_helix_coords from my_old_offset and psi angle
			transformCartesianAndHelicalCoords(my_old_offset, my_old_offset_helix_coords, rot_deg, tilt_deg, psi_deg, CART_TO_HELICAL_COORDS);
			// We do NOT want to accumulate the offsets in the direction along the helix (which is X in the helical coordinate system!)
			// However, when doing helical local searches, we accumulate offsets
			// Do NOT accumulate offsets in 3D classification of helices
			if ( (baseMLO->mymodel.ref_dim == 3) && (! baseMLO->do_skip_align) && (! baseMLO->do_skip_rotate) )
			{
				// TODO: check whether the following lines make sense
				bool do_auto_refine_local_searches = (baseMLO->do_auto_refine) && (baseMLO->sampling.healpix_order >= baseMLO->autosampling_hporder_local_searches);
				bool do_classification_local_searches = (! baseMLO->do_auto_refine) && (baseMLO->mymodel.orientational_prior_mode == PRIOR_ROTTILT_PSI)
						&& (baseMLO->mymodel.sigma2_rot > 0.) && (baseMLO->mymodel.sigma2_tilt > 0.) && (baseMLO->mymodel.sigma2_psi > 0.);
				bool do_local_angular_searches = (do_auto_refine_local_searches) || (do_classification_local_searches);
				if (!do_local_angular_searches)
				{
					if (! cudaMLO->dataIs3D)
						XX(my_old_offset_helix_coords) = 0.;
					else
						ZZ(my_old_offset_helix_coords) = 0.;
				}
			}
			// TODO: Now re-calculate the my_old_offset in the real (or image) system of coordinate (rotate -psi angle)
			transformCartesianAndHelicalCoords(my_old_offset_helix_coords, my_old_offset, rot_deg, tilt_deg, psi_deg, HELICAL_TO_CART_COORDS);
		}


		my_old_offset.selfROUND();
		CTIC(cudaMLO->timer,"kernel_translate");
		if(cudaMLO->dataIs3D)
			cuda_kernel_translate3D<<<STBsize,BLOCK_SIZE>>>(
								~temp,  // translate from temp...
								~d_img, // ... into d_img
								img_size,
								img.data.xdim,
								img.data.ydim,
								img.data.zdim,
								XX(my_old_offset),
								YY(my_old_offset),
								ZZ(my_old_offset));
		else
			cuda_kernel_translate2D<<<STBsize,BLOCK_SIZE>>>(
								~temp,  // translate from temp...
								~d_img, // ... into d_img
								img_size,
								img.data.xdim,
								img.data.ydim,
								XX(my_old_offset),
								YY(my_old_offset));
		LAUNCH_PRIVATE_ERROR(hipGetLastError(),cudaMLO->errorStatus);
		CTOC(cudaMLO->timer,"kernel_translate");

		if (baseMLO->has_converged && baseMLO->do_use_reconstruct_images) //rec_img is NOT norm_corrected in the CPU-code, so nor do we.
		{
			for (int i=0; i<img_size; i++)
				temp[i] = rec_img.data.data[i];
			temp.cp_to_device();
			temp.streamSync();
			if(cudaMLO->dataIs3D)
				cuda_kernel_translate3D<<<STBsize,BLOCK_SIZE>>>(
									~temp,  // translate from temp...
									~d_img, // ... into d_img
									img_size,
									img.data.xdim,
									img.data.ydim,
									img.data.zdim,
									XX(my_old_offset),
									YY(my_old_offset),
									ZZ(my_old_offset));
			else
				cuda_kernel_translate2D<<<STBsize,BLOCK_SIZE>>>(
									~temp,  // translate from temp...
									~d_img, // ... into d_img
									img_size,
									img.data.xdim,
									img.data.ydim,
									XX(my_old_offset),
									YY(my_old_offset));
			LAUNCH_PRIVATE_ERROR(hipGetLastError(),cudaMLO->errorStatus);
		}

		if ( (baseMLO->do_helical_refine) && (! baseMLO->ignore_helical_symmetry) )
		{
			// Transform rounded Cartesian offsets to corresponding helical ones
			transformCartesianAndHelicalCoords(my_old_offset, my_old_offset_helix_coords, rot_deg, tilt_deg, psi_deg, CART_TO_HELICAL_COORDS);
			op.old_offset[ipart] = my_old_offset_helix_coords;
		}
		else
		{
			op.old_offset[ipart] = my_old_offset;  // Not doing helical refinement. Rounded Cartesian offsets are stored.
		}
		// Also store priors on translations
		op.prior[ipart] = my_prior;

		CTOC(cudaMLO->timer,"selfTranslate");

		CTIC(cudaMLO->timer,"calcFimg");
		size_t current_size_x = baseMLO->mymodel.current_size / 2 + 1;
		size_t current_size_y = baseMLO->mymodel.current_size;
		size_t current_size_z = (cudaMLO->dataIs3D) ? baseMLO->mymodel.current_size : 1;

		cudaMLO->transformer1.setSize(img().xdim,img().ydim,img().zdim);

		//FIXME What is this?
//		deviceInitValue(cudaMLO->transformer1.reals, (XFLOAT)0.);
//		deviceInitComplexValue(cudaMLO->transformer1.fouriers, (XFLOAT)0.);
//		cudaMLO->transformer1.reals.streamSync();
//		cudaMLO->transformer1.fouriers.streamSync();

		d_img.cp_on_device(cudaMLO->transformer1.reals);

		runCenterFFT(
				cudaMLO->transformer1.reals,
				(int)cudaMLO->transformer1.xSize,
				(int)cudaMLO->transformer1.ySize,
				(int)cudaMLO->transformer1.zSize,
				false
				);

		cudaMLO->transformer1.reals.streamSync();

		cudaMLO->transformer1.forward();
		cudaMLO->transformer1.fouriers.streamSync();

		int FMultiBsize = ( (int) ceilf(( float)cudaMLO->transformer1.fouriers.getSize()*2/(float)BLOCK_SIZE));
		cuda_kernel_multi<<<FMultiBsize,BLOCK_SIZE,0,cudaMLO->transformer1.fouriers.getStream()>>>(
						(XFLOAT*)~cudaMLO->transformer1.fouriers,
						(XFLOAT)1/((XFLOAT)(cudaMLO->transformer1.reals.getSize())),
						cudaMLO->transformer1.fouriers.getSize()*2);
		LAUNCH_PRIVATE_ERROR(hipGetLastError(),cudaMLO->errorStatus);

		CudaGlobalPtr<CUDACOMPLEX> d_Fimg(current_size_x * current_size_y * current_size_z, cudaMLO->devBundle->allocator);
		d_Fimg.device_alloc();

		cudaMLO->transformer1.fouriers.streamSync();

		windowFourierTransform2(
				cudaMLO->transformer1.fouriers,
				d_Fimg,
				cudaMLO->transformer1.xFSize,cudaMLO->transformer1.yFSize, cudaMLO->transformer1.zFSize, //Input dimensions
				current_size_x, current_size_y, current_size_z  //Output dimensions
				);
		CTOC(cudaMLO->timer,"calcFimg");
		cudaMLO->transformer1.fouriers.streamSync();

		CTIC(cudaMLO->timer,"cpFimg2Host");
		d_Fimg.cp_to_host();
		d_Fimg.streamSync();

		Fimg.initZeros(current_size_z, current_size_y, current_size_x);
		for (int i = 0; i < Fimg.nzyxdim; i ++)
		{
			Fimg.data[i].real = (RFLOAT) d_Fimg[i].x;
			Fimg.data[i].imag = (RFLOAT) d_Fimg[i].y;
		}
		CTOC(cudaMLO->timer,"cpFimg2Host");

		CTIC(cudaMLO->timer,"selfApplyBeamTilt");
		// Here apply the beamtilt correction if necessary
		// This will only be used for reconstruction, not for alignment
		// But beamtilt only affects very high-resolution components anyway...
		//
		RFLOAT beamtilt_x = DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_BEAMTILT_X);
		RFLOAT beamtilt_y = DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_BEAMTILT_Y);
		RFLOAT Cs = DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_CTF_CS);
		RFLOAT V = 1000. * DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_CTF_VOLTAGE);
		RFLOAT lambda = 12.2643247 / sqrt(V * (1. + V * 0.978466e-6));
		if (ABS(beamtilt_x) > 0. || ABS(beamtilt_y) > 0.)
			selfApplyBeamTilt(Fimg, beamtilt_x, beamtilt_y, lambda, Cs,baseMLO->mymodel.pixel_size, baseMLO->mymodel.ori_size);

		op.Fimgs_nomask.at(ipart) = Fimg;

		CTOC(cudaMLO->timer,"selfApplyBeamTilt");

		CTIC(cudaMLO->timer,"zeroMask");
		MultidimArray<RFLOAT> Mnoise;
		bool is_helical_segment = (baseMLO->do_helical_refine) || ((baseMLO->mymodel.ref_dim == 2) && (baseMLO->helical_tube_outer_diameter > 0.));

		if (!baseMLO->do_zero_mask)
		{
			// Make a noisy background image with the same spectrum as the sigma2_noise

			// Different MPI-distributed subsets may otherwise have different instances of the random noise below,
			// because work is on an on-demand basis and therefore variable with the timing of distinct nodes...
			// Have the seed based on the part_id, so that each particle has a different instant of the noise
			if (baseMLO->do_realign_movies)
				init_random_generator(baseMLO->random_seed + part_id);
			else
				init_random_generator(baseMLO->random_seed + my_ori_particle); // This only serves for exact reproducibility tests with 1.3-code...

			// If we're doing running averages, then the sigma2_noise was already adjusted for the running averages.
			// Undo this adjustment here in order to get the right noise in the individual frames
			MultidimArray<RFLOAT> power_noise = baseMLO->sigma2_fudge * baseMLO->mymodel.sigma2_noise[group_id];
			if (baseMLO->do_realign_movies)
				power_noise *= (2. * baseMLO->movie_frame_running_avg_side + 1.);

			// Create noisy image for outside the mask
			MultidimArray<Complex > Fnoise;
			Mnoise.resize(img());
			cudaMLO->transformer.setReal(Mnoise);
			cudaMLO->transformer.getFourierAlias(Fnoise);
			// Fill Fnoise with random numbers, use power spectrum of the noise for its variance
			FOR_ALL_ELEMENTS_IN_FFTW_TRANSFORM(Fnoise)
			{
				int ires = ROUND( sqrt( (RFLOAT)(kp * kp + ip * ip + jp * jp) ) );
				if (ires >= 0 && ires < XSIZE(Fnoise))
				{
					RFLOAT sigma = sqrt(DIRECT_A1D_ELEM(power_noise, ires));
					DIRECT_A3D_ELEM(Fnoise, k, i, j).real = rnd_gaus(0., sigma);
					DIRECT_A3D_ELEM(Fnoise, k, i, j).imag = rnd_gaus(0., sigma);
				}
				else
				{
					DIRECT_A3D_ELEM(Fnoise, k, i, j) = 0.;
				}
			}
			// Back to real space Mnoise
			CTIC(cudaMLO->timer,"inverseFourierTransform");
			cudaMLO->transformer.inverseFourierTransform();
			CTOC(cudaMLO->timer,"inverseFourierTransform");

			CTIC(cudaMLO->timer,"setXmippOrigin2");
			Mnoise.setXmippOrigin();
			CTOC(cudaMLO->timer,"setXmippOrigin2");

			CTIC(cudaMLO->timer,"softMaskOutsideMap");
			d_img.cp_to_host();
			d_img.streamSync();
			for (int i=0; i<img_size; i++)
				img.data.data[i] = d_img[i];

			if (is_helical_segment)
			{
				softMaskOutsideMapForHelix(img(), psi_deg, tilt_deg, (baseMLO->particle_diameter / (2. * baseMLO->mymodel.pixel_size)),
						(baseMLO->helical_tube_outer_diameter / (2. * baseMLO->mymodel.pixel_size)), baseMLO->width_mask_edge, &Mnoise);
			}
			else
				softMaskOutsideMap(img(), baseMLO->particle_diameter / (2. * baseMLO->mymodel.pixel_size), (RFLOAT)baseMLO->width_mask_edge, &Mnoise);

			for (int i=0; i<img_size; i++)
				d_img[i] = img.data.data[i];
			d_img.cp_to_device();

			CTOC(cudaMLO->timer,"softMaskOutsideMap");
		}
		else if (is_helical_segment)
		{
			d_img.cp_to_host();
			d_img.streamSync();
			for (int i=0; i<img_size; i++)
				img.data.data[i] = d_img[i];

			softMaskOutsideMapForHelix(img(), psi_deg, tilt_deg, (baseMLO->particle_diameter / (2. * baseMLO->mymodel.pixel_size)),
					(baseMLO->helical_tube_outer_diameter / (2. * baseMLO->mymodel.pixel_size)), baseMLO->width_mask_edge);

			for (int i=0; i<img_size; i++)
				d_img[i] = img.data.data[i];
			d_img.cp_to_device();
		}
		else
		{
			CTIC(cudaMLO->timer,"softMaskOutsideMap");

			XFLOAT cosine_width = baseMLO->width_mask_edge;
			XFLOAT radius = (XFLOAT)((RFLOAT)baseMLO->particle_diameter / (2. *baseMLO-> mymodel.pixel_size));
			if (radius < 0)
				radius = ((RFLOAT)img.data.xdim)/2.;
			XFLOAT radius_p = radius + cosine_width;

//			dim3 block_dim = 1; //TODO
//			cuda_kernel_softMaskOutsideMap<<<block_dim,SOFTMASK_BLOCK_SIZE>>>(	~d_img,
//																				img().nzyxdim,
//																				img.data.xdim,
//																				img.data.ydim,
//																				img.data.zdim,
//																				img.data.xdim/2,
//																				img.data.ydim/2,
//																				img.data.zdim/2, //unused
//																				true,
//																				radius,
//																				radius_p,
//																				cosine_width);
//			LAUNCH_PRIVATE_ERROR(hipGetLastError(),cudaMLO->errorStatus);

			XFLOAT sum_bg(0.);
			dim3 block_dim = 128; //TODO: set balanced (hardware-dep?)
			CudaGlobalPtr<XFLOAT> softMaskSum   (SOFTMASK_BLOCK_SIZE,0,cudaMLO->devBundle->allocator);
			CudaGlobalPtr<XFLOAT> softMaskSum_bg(SOFTMASK_BLOCK_SIZE,0,cudaMLO->devBundle->allocator);
			softMaskSum.device_alloc();
			softMaskSum_bg.device_alloc();
			softMaskSum.device_init(0.f);
			softMaskSum_bg.device_init(0.f);
			cuda_kernel_softMaskBackgroundValue<<<block_dim,SOFTMASK_BLOCK_SIZE>>>(	~d_img,
																				img().nzyxdim,
																				img.data.xdim,
																				img.data.ydim,
																				img.data.zdim,
																				img.data.xdim/2,
																				img.data.ydim/2,
																				img.data.zdim/2, //unused
																				true,
																				radius,
																				radius_p,
																				cosine_width,
																				~softMaskSum,
																				~softMaskSum_bg);
			LAUNCH_PRIVATE_ERROR(hipGetLastError(),cudaMLO->errorStatus);

			softMaskSum.streamSync();
			sum_bg = (RFLOAT) getSumOnDevice(softMaskSum_bg) / (RFLOAT) getSumOnDevice(softMaskSum);
			softMaskSum.streamSync();

			cuda_kernel_cosineFilter<<<block_dim,SOFTMASK_BLOCK_SIZE>>>(	~d_img,
																			img().nzyxdim,
																			img.data.xdim,
																			img.data.ydim,
																			img.data.zdim,
																			img.data.xdim/2,
																			img.data.ydim/2,
																			img.data.zdim/2, //unused
																			true,
																			radius,
																			radius_p,
																			cosine_width,
																			sum_bg);
			LAUNCH_PRIVATE_ERROR(hipGetLastError(),cudaMLO->errorStatus);

//			d_img.streamSync();
//			d_img.cp_to_host();
			DEBUG_HANDLE_ERROR(hipStreamSynchronize(hipStreamPerThread));

//			FOR_ALL_DIRECT_ELEMENTS_IN_MULTIDIMARRAY(img())
//			{
//				img.data.data[n]=(RFLOAT)d_img[n];
//			}

			CTOC(cudaMLO->timer,"softMaskOutsideMap");
		}
		CTOC(cudaMLO->timer,"zeroMask");

		CTIC(cudaMLO->timer,"setSize");
		cudaMLO->transformer1.setSize(img().xdim,img().ydim,img().zdim);
//		deviceInitValue(cudaMLO->transformer1.reals, (XFLOAT)0.);
//		deviceInitComplexValue(cudaMLO->transformer1.fouriers, (XFLOAT)0.);
//		cudaMLO->transformer1.reals.streamSync();
//		cudaMLO->transformer1.fouriers.streamSync();
		CTOC(cudaMLO->timer,"setSize");

		CTIC(cudaMLO->timer,"transform");
		d_img.cp_on_device(cudaMLO->transformer1.reals);

		runCenterFFT(								// runs on input GlobalPtr.stream
				cudaMLO->transformer1.reals,
				(int)cudaMLO->transformer1.xSize,
				(int)cudaMLO->transformer1.ySize,
				(int)cudaMLO->transformer1.zSize,
				false
				);
		cudaMLO->transformer1.reals.streamSync();
		cudaMLO->transformer1.forward();
		cudaMLO->transformer1.fouriers.streamSync();

		int FMultiBsize2 = ( (int) ceilf(( float)cudaMLO->transformer1.fouriers.getSize()*2/(float)BLOCK_SIZE));
		cuda_kernel_multi<<<FMultiBsize2,BLOCK_SIZE,0,cudaMLO->transformer1.fouriers.getStream()>>>(
						(XFLOAT*)~cudaMLO->transformer1.fouriers,
						(XFLOAT)1/((XFLOAT)(cudaMLO->transformer1.reals.getSize())),
						cudaMLO->transformer1.fouriers.getSize()*2);
		LAUNCH_PRIVATE_ERROR(hipGetLastError(),cudaMLO->errorStatus);

		CTOC(cudaMLO->timer,"transform");

		cudaMLO->transformer1.fouriers.streamSync();

		CTIC(cudaMLO->timer,"powerClass");
		// Store the power_class spectrum of the whole image (to fill sigma2_noise between current_size and ori_size
		if (baseMLO->mymodel.current_size < baseMLO->mymodel.ori_size)
		{
			CudaGlobalPtr<XFLOAT> spectrumAndXi2((baseMLO->mymodel.ori_size/2+1)+1,0,cudaMLO->devBundle->allocator); // last +1 is the Xi2, to remove an expensive memcpy
			spectrumAndXi2.device_alloc();
			spectrumAndXi2.device_init(0);
			spectrumAndXi2.streamSync();

			dim3 gridSize = CEIL((float)(cudaMLO->transformer1.fouriers.getSize()) / (float)POWERCLASS_BLOCK_SIZE);
			if(cudaMLO->dataIs3D)
				cuda_kernel_powerClass<true><<<gridSize,POWERCLASS_BLOCK_SIZE,0,0>>>(
					~cudaMLO->transformer1.fouriers,
					~spectrumAndXi2,
					cudaMLO->transformer1.fouriers.getSize(),
					spectrumAndXi2.getSize()-1,
					cudaMLO->transformer1.xFSize,
					cudaMLO->transformer1.yFSize,
					cudaMLO->transformer1.zFSize,
					(baseMLO->mymodel.current_size/2)+1, // note: NOT baseMLO->mymodel.ori_size/2+1
					&spectrumAndXi2.d_ptr[spectrumAndXi2.getSize()-1]); // last element is the hihgres_Xi2
			else
				cuda_kernel_powerClass<false><<<gridSize,POWERCLASS_BLOCK_SIZE,0,0>>>(
					~cudaMLO->transformer1.fouriers,
					~spectrumAndXi2,
					cudaMLO->transformer1.fouriers.getSize(),
					spectrumAndXi2.getSize()-1,
					cudaMLO->transformer1.xFSize,
					cudaMLO->transformer1.yFSize,
					cudaMLO->transformer1.zFSize,
					(baseMLO->mymodel.current_size/2)+1, // note: NOT baseMLO->mymodel.ori_size/2+1
					&spectrumAndXi2.d_ptr[spectrumAndXi2.getSize()-1]); // last element is the hihgres_Xi2

			LAUNCH_PRIVATE_ERROR(hipGetLastError(),cudaMLO->errorStatus);

			spectrumAndXi2.streamSync();
			spectrumAndXi2.cp_to_host();
			spectrumAndXi2.streamSync();

			op.power_imgs.at(ipart).resize(baseMLO->mymodel.ori_size/2 + 1);

			for (int i = 0; i<(spectrumAndXi2.getSize()-1); i ++)
				op.power_imgs.at(ipart).data[i] = spectrumAndXi2[i];
			op.highres_Xi2_imgs.at(ipart) = spectrumAndXi2[spectrumAndXi2.getSize()-1];
		}
		else
		{
			op.highres_Xi2_imgs.at(ipart) = 0.;
		}
		CTOC(cudaMLO->timer,"powerClass");
		// We never need any resolutions higher than current_size
		// So resize the Fourier transforms
		CTIC(cudaMLO->timer,"windowFourierTransform2");
		//windowFourierTransform(Faux, Fimg, baseMLO->mymodel.current_size);
		cudaMLO->transformer1.fouriers.streamSync();
		windowFourierTransform2(
				cudaMLO->transformer1.fouriers,
				d_Fimg,
				cudaMLO->transformer1.xFSize,cudaMLO->transformer1.yFSize, cudaMLO->transformer1.zFSize, //Input dimensions
				current_size_x, current_size_y, current_size_z,  //Output dimensions
				1, 	//Npsi
				0,	//pos
				cudaMLO->transformer1.fouriers.getStream()
				);
		CTOC(cudaMLO->timer,"windowFourierTransform2");
		// Also store its CTF
		CTIC(cudaMLO->timer,"ctfCorr");
		CTIC(cudaMLO->timer,"cpFimg2Host_2");
		d_Fimg.streamSync();
		d_Fimg.cp_to_host();
		d_Fimg.streamSync();
		for (int i = 0; i < Fimg.nzyxdim; i ++)
		{
			Fimg.data[i].real = (RFLOAT) d_Fimg[i].x;
			Fimg.data[i].imag = (RFLOAT) d_Fimg[i].y;
		}
		CTOC(cudaMLO->timer,"cpFimg2Host_2");

		Fctf.resize(Fimg);
		// Now calculate the actual CTF
		if (baseMLO->do_ctf_correction)
		{
			if (cudaMLO->dataIs3D)
			{
				Image<RFLOAT> Ictf;
				if (baseMLO->do_parallel_disc_io)
				{
					CTIC(cudaMLO->timer,"CTFRead3D_disk");
					// Read CTF-image from disc
					FileName fn_ctf;
					if (!baseMLO->mydata.getImageNameOnScratch(part_id, fn_ctf, true))
					{
						std::istringstream split(baseMLO->exp_fn_ctf);
						// Get the right line in the exp_fn_img string
						for (int i = 0; i <= istop; i++)
							getline(split, fn_ctf);
					}
					Ictf.read(fn_ctf);
					CTOC(cudaMLO->timer,"CTFRead3D_disk");
				}
				else
				{
					CTIC(cudaMLO->timer,"CTFRead3D_array");
					// Unpack the CTF-image from the exp_imagedata array
					Ictf().resize(baseMLO->mymodel.ori_size, baseMLO->mymodel.ori_size, baseMLO->mymodel.ori_size);
					FOR_ALL_DIRECT_ELEMENTS_IN_ARRAY3D(Ictf())
					{
						DIRECT_A3D_ELEM(Ictf(), k, i, j) = DIRECT_A3D_ELEM(baseMLO->exp_imagedata, baseMLO->mymodel.ori_size + k, i, j);
					}
					CTOC(cudaMLO->timer,"CTFRead3D_array");
				}
				// Set the CTF-image in Fctf
				CTIC(cudaMLO->timer,"CTFSet3D_array");
				Ictf().setXmippOrigin();
				FOR_ALL_ELEMENTS_IN_FFTW_TRANSFORM(Fctf)
				{
					// Use negative kp,ip and jp indices, because the origin in the ctf_img lies half a pixel to the right of the actual center....
					DIRECT_A3D_ELEM(Fctf, k, i, j) = A3D_ELEM(Ictf(), -kp, -ip, -jp);
				}
				CTIC(cudaMLO->timer,"CTFSet3D_array");
			}
			else
			{
				CTIC(cudaMLO->timer,"CTFRead2D");
				CTF ctf;
				ctf.setValues(DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_CTF_DEFOCUS_U),
							  DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_CTF_DEFOCUS_V),
							  DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_CTF_DEFOCUS_ANGLE),
							  DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_CTF_VOLTAGE),
							  DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_CTF_CS),
							  DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_CTF_Q0),
							  DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_CTF_BFAC),
                                                          1.,
                                                          DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_CTF_PHASE_SHIFT));

				ctf.getFftwImage(Fctf, baseMLO->mymodel.ori_size, baseMLO->mymodel.ori_size, baseMLO->mymodel.pixel_size,
						baseMLO->ctf_phase_flipped, baseMLO->only_flip_phases, baseMLO->intact_ctf_first_peak, true);
				CTIC(cudaMLO->timer,"CTFRead2D");
			}
		}
		else
		{
			Fctf.initConstant(1.);
		}
		CTOC(cudaMLO->timer,"ctfCorr");
		// Store Fimg and Fctf
		op.Fimgs.at(ipart) = Fimg;
		op.Fctfs.at(ipart) = Fctf;

	} // end loop ipart
	//cudaMLO->transformer.clear();
#ifdef TIMING
	if (op.my_ori_particle == baseMLO->exp_my_first_ori_particle)
		baseMLO->timer.toc(baseMLO->TIMING_ESP_FT);
#endif
	GTOC(cudaMLO->timer,"getFourierTransformsAndCtfs");
	GATHERGPUTIMINGS(cudaMLO->timer);
}

void getAllSquaredDifferencesCoarse(
		unsigned exp_ipass,
		OptimisationParamters &op,
		SamplingParameters &sp,
		MlOptimiser *baseMLO,
		MlOptimiserCuda *cudaMLO,
	 	CudaGlobalPtr<XFLOAT> &Mweight)
{

#ifdef TIMING
	if (op.my_ori_particle == baseMLO->exp_my_first_ori_particle)
		baseMLO->timer.tic(baseMLO->TIMING_ESP_DIFF1);
#endif

	CUSTOM_ALLOCATOR_REGION_NAME("DIFF_COARSE");

	CTIC(cudaMLO->timer,"diff_pre_gpu");
	unsigned long weightsPerPart(baseMLO->mymodel.nr_classes * sp.nr_dir * sp.nr_psi * sp.nr_trans * sp.nr_oversampled_rot * sp.nr_oversampled_trans);

	std::vector<MultidimArray<Complex > > dummy;
	baseMLO->precalculateShiftedImagesCtfsAndInvSigma2s(false, op.my_ori_particle, sp.current_image_size, sp.current_oversampling, op.metadata_offset, // inserted SHWS 12112015
			sp.itrans_min, sp.itrans_max, op.Fimgs, dummy, op.Fctfs, dummy, dummy,
			op.local_Fctfs, op.local_sqrtXi2, op.local_Minvsigma2s);

	unsigned image_size = op.local_Minvsigma2s[0].nzyxdim;

	CTOC(cudaMLO->timer,"diff_pre_gpu");

	std::vector<CudaProjectorPlan> projectorPlans(0, cudaMLO->devBundle->allocator);

	//If particle specific sampling plan required
	if (cudaMLO->devBundle->generateProjectionPlanOnTheFly)
	{
		CTIC(cudaMLO->timer,"generateProjectionSetupCoarse");

		projectorPlans.resize(baseMLO->mymodel.nr_classes, cudaMLO->devBundle->allocator);

		for (int iclass = sp.iclass_min; iclass <= sp.iclass_max; iclass++)
		{
			if (baseMLO->mymodel.pdf_class[iclass] > 0.)
			{
				projectorPlans[iclass].setup(
						baseMLO->sampling,
						op.directions_prior,
						op.psi_prior,
						op.pointer_dir_nonzeroprior,
						op.pointer_psi_nonzeroprior,
						NULL, //Mcoarse_significant
						baseMLO->mymodel.pdf_class,
						baseMLO->mymodel.pdf_direction,
						sp.nr_dir,
						sp.nr_psi,
						sp.idir_min,
						sp.idir_max,
						sp.ipsi_min,
						sp.ipsi_max,
						sp.itrans_min,
						sp.itrans_max,
						0, //current_oversampling
						1, //nr_oversampled_rot
						iclass,
						true, //coarse
						!IS_NOT_INV,
						baseMLO->do_skip_align,
						baseMLO->do_skip_rotate,
						baseMLO->mymodel.orientational_prior_mode
						);
			}
		}
		CTOC(cudaMLO->timer,"generateProjectionSetupCoarse");
	}
	else
		projectorPlans = cudaMLO->devBundle->coarseProjectionPlans;

	// Loop only from sp.iclass_min to sp.iclass_max to deal with seed generation in first iteration
	size_t allWeights_size(0);
	for (int exp_iclass = sp.iclass_min; exp_iclass <= sp.iclass_max; exp_iclass++)
		allWeights_size += projectorPlans[exp_iclass].orientation_num * sp.nr_trans*sp.nr_oversampled_trans;

	CudaGlobalPtr<XFLOAT> allWeights(allWeights_size,cudaMLO->devBundle->allocator);
	allWeights.device_alloc();

	long int allWeights_pos=0;	bool do_CC = (baseMLO->iter == 1 && baseMLO->do_firstiter_cc) || baseMLO->do_always_cc;

	for (long int ipart = 0; ipart < sp.nr_particles; ipart++)
	{
		long int part_id = baseMLO->mydata.ori_particles[op.my_ori_particle].particles_id[ipart];
		long int group_id = baseMLO->mydata.getGroupId(part_id);

		/*====================================
				Generate Translations
		======================================*/

		CTIC(cudaMLO->timer,"translation_1");

		long unsigned translation_num((sp.itrans_max - sp.itrans_min + 1) * sp.nr_oversampled_trans);

		CudaGlobalPtr<XFLOAT> trans_x(translation_num, cudaMLO->devBundle->allocator);
		CudaGlobalPtr<XFLOAT> trans_y(translation_num, cudaMLO->devBundle->allocator);
		CudaGlobalPtr<XFLOAT> trans_z(translation_num, cudaMLO->devBundle->allocator);

		CudaGlobalPtr<XFLOAT> Fimg_real(image_size, cudaMLO->devBundle->allocator);
		CudaGlobalPtr<XFLOAT> Fimg_imag(image_size, cudaMLO->devBundle->allocator);

		std::vector<RFLOAT> oversampled_translations_x, oversampled_translations_y, oversampled_translations_z;

		for (long int itrans = 0; itrans < translation_num; itrans++)
		{
			baseMLO->sampling.getTranslations(itrans, 0, oversampled_translations_x,
					oversampled_translations_y, oversampled_translations_z,
					(baseMLO->do_helical_refine) && (! baseMLO->ignore_helical_symmetry), baseMLO->helical_rise_initial / baseMLO->mymodel.pixel_size, baseMLO->helical_twist_initial);

			RFLOAT xshift = 0., yshift = 0., zshift = 0.;

			xshift = oversampled_translations_x[0];
			yshift = oversampled_translations_y[0];
			if (cudaMLO->dataIs3D)
				zshift = oversampled_translations_z[0];

			if ( (baseMLO->do_helical_refine) && (! baseMLO->ignore_helical_symmetry) )
			{
				RFLOAT rot_deg = DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_ROT);
				RFLOAT tilt_deg = DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_TILT);
				RFLOAT psi_deg = DIRECT_A2D_ELEM(baseMLO->exp_metadata,op.metadata_offset + ipart, METADATA_PSI);
				transformCartesianAndHelicalCoords(xshift, yshift, zshift, xshift, yshift, zshift, rot_deg, tilt_deg, psi_deg, (cudaMLO->dataIs3D) ? (3) : (2), HELICAL_TO_CART_COORDS);
			}

			trans_x[itrans] = -2 * PI * xshift / (double)baseMLO->mymodel.ori_size;
			trans_y[itrans] = -2 * PI * yshift / (double)baseMLO->mymodel.ori_size;
			trans_z[itrans] = -2 * PI * zshift / (double)baseMLO->mymodel.ori_size;
		}

		XFLOAT scale_correction = baseMLO->do_scale_correction ? baseMLO->mymodel.scale_correction[group_id] : 1;

		MultidimArray<Complex > Fimg;
		windowFourierTransform(op.Fimgs[ipart], Fimg, sp.current_image_size);

		for (unsigned i = 0; i < image_size; i ++)
		{
			XFLOAT pixel_correction = 1.0/scale_correction;
			if (baseMLO->do_ctf_correction && baseMLO->refs_are_ctf_corrected)
			{
				// if ctf[i]==0, pix_corr[i] becomes NaN.
				// However, corr_img[i]==0, so pix-diff in kernel==0.
				// This is ok since originally, pix-diff==Img.real^2 + Img.imag^2,
				// which is ori-indep, and we subtract min_diff form ALL orients.
				if (op.local_Fctfs[ipart].data[i]!=0)
					pixel_correction /= op.local_Fctfs[ipart].data[i];
			}
			Fimg_real[i] = Fimg.data[i].real * pixel_correction;
			Fimg_imag[i] = Fimg.data[i].imag * pixel_correction;
		}

		trans_x.put_on_device();
		trans_y.put_on_device();
		trans_z.put_on_device();

		Fimg_real.put_on_device();
		Fimg_imag.put_on_device();

		CTOC(cudaMLO->timer,"translation_1");

		// To speed up calculation, several image-corrections are grouped into a single pixel-wise "filter", or image-correciton
		CudaGlobalPtr<XFLOAT> corr_img(image_size, cudaMLO->devBundle->allocator);
		corr_img.device_alloc();

		buildCorrImage(baseMLO,op,corr_img,ipart,group_id);
		corr_img.cp_to_device();

		deviceInitValue(allWeights, (XFLOAT) (op.highres_Xi2_imgs[ipart] / 2.));
		allWeights_pos = 0;

		for (int exp_iclass = sp.iclass_min; exp_iclass <= sp.iclass_max; exp_iclass++)
			DEBUG_HANDLE_ERROR(hipStreamSynchronize(cudaMLO->classStreams[exp_iclass]));
		DEBUG_HANDLE_ERROR(hipStreamSynchronize(hipStreamPerThread));

		for (int exp_iclass = sp.iclass_min; exp_iclass <= sp.iclass_max; exp_iclass++)
		{
			if ( projectorPlans[exp_iclass].orientation_num > 0 )
			{
				/*====================================
				    	   Kernel Call
				======================================*/

				CudaProjectorKernel projKernel = CudaProjectorKernel::makeKernel(
						cudaMLO->devBundle->cudaProjectors[exp_iclass],
						op.local_Minvsigma2s[0].xdim,
						op.local_Minvsigma2s[0].ydim,
						op.local_Minvsigma2s[0].zdim,
						op.local_Minvsigma2s[0].xdim-1);

				runDiff2KernelCoarse(
						projKernel,
						~trans_x,
						~trans_y,
						~trans_z,
						~corr_img,
						~Fimg_real,
						~Fimg_imag,
						~projectorPlans[exp_iclass].eulers,
						&allWeights(allWeights_pos),
						(XFLOAT) op.local_sqrtXi2[ipart],
						projectorPlans[exp_iclass].orientation_num,
						translation_num,
						image_size,
						cudaMLO->classStreams[exp_iclass],
						do_CC,
						cudaMLO->dataIs3D);

				mapAllWeightsToMweights(
						~projectorPlans[exp_iclass].iorientclasses,
						&allWeights(allWeights_pos),
						&Mweight(ipart*weightsPerPart),
						projectorPlans[exp_iclass].orientation_num,
						translation_num,
						cudaMLO->classStreams[exp_iclass]
						);

				/*====================================
				    	   Retrieve Results
				======================================*/
				allWeights_pos += projectorPlans[exp_iclass].orientation_num*translation_num;

			}
		}

		for (int exp_iclass = sp.iclass_min; exp_iclass <= sp.iclass_max; exp_iclass++)
			DEBUG_HANDLE_ERROR(hipStreamSynchronize(cudaMLO->classStreams[exp_iclass]));
		DEBUG_HANDLE_ERROR(hipStreamSynchronize(hipStreamPerThread)); // does not appear to be NEEDED FOR NON-BLOCKING CLASS STREAMS in tests, but should be to sync against classStreams

		op.min_diff2[ipart] = getMinOnDevice(allWeights);
		op.avg_diff2[ipart] = (RFLOAT) getSumOnDevice(allWeights) / (RFLOAT) allWeights_size;

	} // end loop ipart

#ifdef TIMING
	if (op.my_ori_particle == baseMLO->exp_my_first_ori_particle)
		baseMLO->timer.toc(baseMLO->TIMING_ESP_DIFF1);
#endif
}

void getAllSquaredDifferencesFine(unsigned exp_ipass,
		 	 	 	 	 	 	  OptimisationParamters &op,
		 	 	 	 	 	 	  SamplingParameters &sp,
		 	 	 	 	 	 	  MlOptimiser *baseMLO,
		 	 	 	 	 	 	  MlOptimiserCuda *cudaMLO,
		 	 	 	 	 	 	  std::vector<IndexedDataArray> &FinePassWeights,
		 	 	 	 	 	 	  std::vector<std::vector< IndexedDataArrayMask > > &FPCMasks,
		 	 	 	 	 	 	  std::vector<ProjectionParams> &FineProjectionData,
		 	 	 	 	 	 	  std::vector<cudaStager<unsigned long> > &stagerD2)
{
#ifdef TIMING
	if (op.my_ori_particle == baseMLO->exp_my_first_ori_particle)
		baseMLO->timer.tic(baseMLO->TIMING_ESP_DIFF2);
#endif

	CUSTOM_ALLOCATOR_REGION_NAME("DIFF_FINE");
	CTIC(cudaMLO->timer,"diff_pre_gpu");

	CTIC(cudaMLO->timer,"precalculateShiftedImagesCtfsAndInvSigma2s");
	std::vector<MultidimArray<Complex > > dummy;
	baseMLO->precalculateShiftedImagesCtfsAndInvSigma2s(false, op.my_ori_particle, sp.current_image_size, sp.current_oversampling, op.metadata_offset, // inserted SHWS 12112015
			sp.itrans_min, sp.itrans_max, op.Fimgs, dummy, op.Fctfs, dummy, dummy,
			op.local_Fctfs, op.local_sqrtXi2, op.local_Minvsigma2s);
	CTOC(cudaMLO->timer,"precalculateShiftedImagesCtfsAndInvSigma2s");
	MultidimArray<Complex > Fref;
	Fref.resize(op.local_Minvsigma2s[0]);

	unsigned image_size = op.local_Minvsigma2s[0].nzyxdim;

	CTOC(cudaMLO->timer,"diff_pre_gpu");

	/*=======================================================================================
										  Particle Iteration
	=========================================================================================*/
	for (long int ipart = 0; ipart < sp.nr_particles; ipart++)
	{
		// Reset size without de-allocating: we will append everything significant within
		// the current allocation and then re-allocate the then determined (smaller) volume

		long int part_id = baseMLO->mydata.ori_particles[op.my_ori_particle].particles_id[ipart];
		long int group_id = baseMLO->mydata.getGroupId(part_id);

		/*====================================
				Generate Translations
		======================================*/

		CTIC(cudaMLO->timer,"translation_2");

		long unsigned translation_num((sp.itrans_max - sp.itrans_min + 1) * sp.nr_oversampled_trans);

		CudaGlobalPtr<XFLOAT> Fimg_real(image_size, cudaMLO->devBundle->allocator);
		CudaGlobalPtr<XFLOAT> Fimg_imag(image_size, cudaMLO->devBundle->allocator);

		CudaGlobalPtr<XFLOAT> trans_x(translation_num, cudaMLO->devBundle->allocator);
		CudaGlobalPtr<XFLOAT> trans_y(translation_num, cudaMLO->devBundle->allocator);
		CudaGlobalPtr<XFLOAT> trans_z(translation_num, cudaMLO->devBundle->allocator);


		std::vector<RFLOAT> oversampled_translations_x, oversampled_translations_y, oversampled_translations_z;

		int j = 0;
		for (long int itrans = 0; itrans < (sp.itrans_max - sp.itrans_min + 1); itrans++)
		{
			baseMLO->sampling.getTranslations(itrans, baseMLO->adaptive_oversampling, oversampled_translations_x,
					oversampled_translations_y, oversampled_translations_z,
					(baseMLO->do_helical_refine) && (! baseMLO->ignore_helical_symmetry), baseMLO->helical_rise_initial / baseMLO->mymodel.pixel_size, baseMLO->helical_twist_initial);

			for (long int iover_trans = 0; iover_trans < oversampled_translations_x.size(); iover_trans++)
			{
				RFLOAT xshift = 0., yshift = 0., zshift = 0.;

				xshift = oversampled_translations_x[iover_trans];
				yshift = oversampled_translations_y[iover_trans];
				if (cudaMLO->dataIs3D)
					zshift = oversampled_translations_z[iover_trans];

				if ( (baseMLO->do_helical_refine) && (! baseMLO->ignore_helical_symmetry) )
				{
					RFLOAT rot_deg = DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_ROT);
					RFLOAT tilt_deg = DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_TILT);
					RFLOAT psi_deg = DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_PSI);
					transformCartesianAndHelicalCoords(xshift, yshift, zshift, xshift, yshift, zshift, rot_deg, tilt_deg, psi_deg, (cudaMLO->dataIs3D) ? (3) : (2), HELICAL_TO_CART_COORDS);
				}

				trans_x[j] = -2 * PI * xshift / (double)baseMLO->mymodel.ori_size;
				trans_y[j] = -2 * PI * yshift / (double)baseMLO->mymodel.ori_size;
				trans_z[j] = -2 * PI * zshift / (double)baseMLO->mymodel.ori_size;
				j ++;
			}
		}

		XFLOAT scale_correction = baseMLO->do_scale_correction ? baseMLO->mymodel.scale_correction[group_id] : 1;

		MultidimArray<Complex > Fimg, Fimg_nomask;
		windowFourierTransform(op.Fimgs[ipart], Fimg, sp.current_image_size);

		for (unsigned i = 0; i < image_size; i ++)
		{
			XFLOAT pixel_correction = 1.0/scale_correction;
			if (baseMLO->do_ctf_correction && baseMLO->refs_are_ctf_corrected)
			{
				// if ctf[i]==0, pix_corr[i] becomes NaN.
				// However, corr_img[i]==0, so pix-diff in kernel==0.
				// This is ok since originally, pix-diff==Img.real^2 + Img.imag^2,
				// which is ori-indep, and we subtract min_diff form ALL orients.
				if (op.local_Fctfs[ipart].data[i]!=0)
					pixel_correction /= op.local_Fctfs[ipart].data[i];
			}

			Fimg_real[i] = Fimg.data[i].real * pixel_correction;
			Fimg_imag[i] = Fimg.data[i].imag * pixel_correction;
		}

		CTOC(cudaMLO->timer,"translation_2");


		CTIC(cudaMLO->timer,"kernel_init_1");

		CudaGlobalPtr<XFLOAT> corr_img(image_size, cudaMLO->devBundle->allocator);
		corr_img.device_alloc();
		buildCorrImage(baseMLO,op,corr_img,ipart,group_id);

		trans_x.put_on_device();
		trans_y.put_on_device();
		trans_z.put_on_device();


		Fimg_real.put_on_device();
		Fimg_imag.put_on_device();
		corr_img.cp_to_device();

		CTOC(cudaMLO->timer,"kernel_init_1");
		std::vector< CudaGlobalPtr<XFLOAT> > eulers((sp.iclass_max-sp.iclass_min+1), cudaMLO->devBundle->allocator);
		cudaStager<XFLOAT> AllEulers(cudaMLO->devBundle->allocator,9*FineProjectionData[ipart].orientationNumAllClasses);
		AllEulers.prepare_device();
		unsigned long newDataSize(0);

		for (int exp_iclass = sp.iclass_min; exp_iclass <= sp.iclass_max; exp_iclass++)
		{
			FPCMasks[ipart][exp_iclass].weightNum=0;

			if ((baseMLO->mymodel.pdf_class[exp_iclass] > 0.) && (FineProjectionData[ipart].class_entries[exp_iclass] > 0) )
			{
				// use "slice" constructor with class-specific parameters to retrieve a temporary ProjectionParams with data for this class
				ProjectionParams thisClassProjectionData(	FineProjectionData[ipart],
															FineProjectionData[ipart].class_idx[exp_iclass],
															FineProjectionData[ipart].class_idx[exp_iclass]+FineProjectionData[ipart].class_entries[exp_iclass]);
				// since we retrieved the ProjectionParams for *the whole* class the orientation_num is also equal.

				thisClassProjectionData.orientation_num[0] = FineProjectionData[ipart].class_entries[exp_iclass];
				long unsigned orientation_num  = thisClassProjectionData.orientation_num[0];

				if(orientation_num==0)
					continue;

				CTIC(cudaMLO->timer,"pair_list_1");
				long unsigned significant_num(0);
				long int nr_over_orient = baseMLO->sampling.oversamplingFactorOrientations(sp.current_oversampling);
				long int nr_over_trans = baseMLO->sampling.oversamplingFactorTranslations(sp.current_oversampling);
				// Prepare the mask of the weight-array for this class
				if (FPCMasks[ipart][exp_iclass].weightNum==0)
					FPCMasks[ipart][exp_iclass].firstPos = newDataSize;

				long unsigned ihidden(0);
				std::vector< long unsigned > iover_transes, ihiddens;

				for (long int itrans = sp.itrans_min; itrans <= sp.itrans_max; itrans++, ihidden++)
				{
					for (long int iover_trans = 0; iover_trans < sp.nr_oversampled_trans; iover_trans++)
					{
						ihiddens.push_back(ihidden);
						iover_transes.push_back(iover_trans);
					}
				}

				int chunkSize(0);
				if(cudaMLO->dataIs3D)
					chunkSize = D2F_CHUNK_DATA3D;
				else if(cudaMLO->refIs3D)
					chunkSize = D2F_CHUNK_DATA3D;
				else
					chunkSize = D2F_CHUNK_2D;

				// Do more significance checks on translations and create jobDivision
				significant_num = makeJobsForDiff2Fine(	op,	sp,												// alot of different type inputs...
														orientation_num, translation_num,
														thisClassProjectionData,
														iover_transes, ihiddens,
														nr_over_orient, nr_over_trans, ipart,
														FinePassWeights[ipart],
														FPCMasks[ipart][exp_iclass],   // ..and output into index-arrays mask...
														chunkSize);                    // ..based on a given maximum chunk-size

				// extend size by number of significants found this class
				newDataSize += significant_num;
				FPCMasks[ipart][exp_iclass].weightNum = significant_num;
				FPCMasks[ipart][exp_iclass].lastPos = FPCMasks[ipart][exp_iclass].firstPos + significant_num;
				CTOC(cudaMLO->timer,"pair_list_1");

				CTIC(cudaMLO->timer,"IndexedArrayMemCp2");
//				FPCMasks[ipart][exp_iclass].jobOrigin.cp_to_device();
//				FPCMasks[ipart][exp_iclass].jobExtent.cp_to_device();
				stagerD2[ipart].stage(FPCMasks[ipart][exp_iclass].jobOrigin);
				stagerD2[ipart].stage(FPCMasks[ipart][exp_iclass].jobExtent);
				CTOC(cudaMLO->timer,"IndexedArrayMemCp2");

				CTIC(cudaMLO->timer,"generateEulerMatrices");
				eulers[exp_iclass-sp.iclass_min].setSize(9*FineProjectionData[ipart].class_entries[exp_iclass]);
				eulers[exp_iclass-sp.iclass_min].host_alloc();
				generateEulerMatrices(
						baseMLO->mymodel.PPref[exp_iclass].padding_factor,
						thisClassProjectionData,
						&(eulers[exp_iclass-sp.iclass_min])[0],
						!IS_NOT_INV);
				AllEulers.stage(eulers[exp_iclass-sp.iclass_min]);
				CTOC(cudaMLO->timer,"generateEulerMatrices");
			}
		}

		// copy stagers to device
		stagerD2[ipart].cp_to_device();
		AllEulers.cp_to_device();

		FinePassWeights[ipart].rot_id.cp_to_device(); //FIXME this is not used
		FinePassWeights[ipart].rot_idx.cp_to_device();
		FinePassWeights[ipart].trans_idx.cp_to_device();

		for (int exp_iclass = sp.iclass_min; exp_iclass <= sp.iclass_max; exp_iclass++)
			DEBUG_HANDLE_ERROR(hipStreamSynchronize(cudaMLO->classStreams[exp_iclass]));
		DEBUG_HANDLE_ERROR(hipStreamSynchronize(hipStreamPerThread));

		for (int exp_iclass = sp.iclass_min; exp_iclass <= sp.iclass_max; exp_iclass++)
		{
			if ((baseMLO->mymodel.pdf_class[exp_iclass] > 0.) && (FineProjectionData[ipart].class_entries[exp_iclass] > 0) )
			{
				long unsigned orientation_num  = FineProjectionData[ipart].class_entries[exp_iclass];
				if(orientation_num==0)
					continue;

				long unsigned significant_num(FPCMasks[ipart][exp_iclass].weightNum);
				if(significant_num==0)
					continue;

				CTIC(cudaMLO->timer,"Diff2MakeKernel");
				CudaProjectorKernel projKernel = CudaProjectorKernel::makeKernel(
						cudaMLO->devBundle->cudaProjectors[exp_iclass],
						op.local_Minvsigma2s[0].xdim,
						op.local_Minvsigma2s[0].ydim,
						op.local_Minvsigma2s[0].zdim,
						op.local_Minvsigma2s[0].xdim-1);
				CTOC(cudaMLO->timer,"Diff2MakeKernel");

				// Use the constructed mask to construct a partial class-specific input
				IndexedDataArray thisClassFinePassWeights(FinePassWeights[ipart],FPCMasks[ipart][exp_iclass], cudaMLO->devBundle->allocator);

				CTIC(cudaMLO->timer,"Diff2CALL");

				runDiff2KernelFine(
						projKernel,
						~corr_img,
						~Fimg_real,
						~Fimg_imag,
						~trans_x,
						~trans_y,
						~trans_z,
						~eulers[exp_iclass-sp.iclass_min],
						~thisClassFinePassWeights.rot_id,
						~thisClassFinePassWeights.rot_idx,
						~thisClassFinePassWeights.trans_idx,
						~FPCMasks[ipart][exp_iclass].jobOrigin,
						~FPCMasks[ipart][exp_iclass].jobExtent,
						~thisClassFinePassWeights.weights,
						op,
						baseMLO,
						orientation_num,
						translation_num,
						significant_num,
						image_size,
						ipart,
						exp_iclass,
						cudaMLO->classStreams[exp_iclass],
						FPCMasks[ipart][exp_iclass].jobOrigin.getSize(),
						((baseMLO->iter == 1 && baseMLO->do_firstiter_cc) || baseMLO->do_always_cc),
						cudaMLO->dataIs3D
						);

//				DEBUG_HANDLE_ERROR(hipStreamSynchronize(hipStreamPerThread));
				CTOC(cudaMLO->timer,"Diff2CALL");

			} // end if class significant
		} // end loop iclass

		for (int exp_iclass = sp.iclass_min; exp_iclass <= sp.iclass_max; exp_iclass++)
			DEBUG_HANDLE_ERROR(hipStreamSynchronize(cudaMLO->classStreams[exp_iclass]));
		DEBUG_HANDLE_ERROR(hipStreamSynchronize(hipStreamPerThread));

		FinePassWeights[ipart].setDataSize( newDataSize );

		CTIC(cudaMLO->timer,"collect_data_1");
		if(baseMLO->adaptive_oversampling!=0)
		{
			op.min_diff2[ipart] = (RFLOAT) getMinOnDevice(FinePassWeights[ipart].weights);
			op.avg_diff2[ipart] = (RFLOAT) getSumOnDevice(FinePassWeights[ipart].weights) /
					(RFLOAT) FinePassWeights[ipart].weights.size;
		}
		CTOC(cudaMLO->timer,"collect_data_1");
//		std::cerr << "  fine pass minweight  =  " << op.min_diff2[ipart] << std::endl;

	}// end loop ipart
#ifdef TIMING
	if (op.my_ori_particle == baseMLO->exp_my_first_ori_particle)
		baseMLO->timer.toc(baseMLO->TIMING_ESP_DIFF2);
#endif
}


template<typename weights_t>
void convertAllSquaredDifferencesToWeights(unsigned exp_ipass,
											OptimisationParamters &op,
											SamplingParameters &sp,
											MlOptimiser *baseMLO,
											MlOptimiserCuda *cudaMLO,
											std::vector< IndexedDataArray> &PassWeights,
											std::vector< std::vector< IndexedDataArrayMask > > &FPCMasks,
											CudaGlobalPtr<XFLOAT> &Mweight, // FPCMasks = Fine-Pass Class-Masks
											bool failsafeMode = false)
{
#ifdef TIMING
	if (op.my_ori_particle == baseMLO->exp_my_first_ori_particle)
	{
		if (exp_ipass == 0) baseMLO->timer.tic(baseMLO->TIMING_ESP_WEIGHT1);
		else baseMLO->timer.tic(baseMLO->TIMING_ESP_WEIGHT2);
	}
#endif

	// Ready the "prior-containers" for all classes (remake every ipart)
	CudaGlobalPtr<XFLOAT>  pdf_orientation((sp.iclass_max-sp.iclass_min+1) * sp.nr_dir * sp.nr_psi, cudaMLO->devBundle->allocator);
	CudaGlobalPtr<XFLOAT>  pdf_offset((sp.iclass_max-sp.iclass_min+1)*sp.nr_trans, cudaMLO->devBundle->allocator);

	RFLOAT pdf_orientation_mean(0);
	unsigned pdf_orientation_count(0);

	CUSTOM_ALLOCATOR_REGION_NAME("CASDTW_PDF");

	pdf_orientation.device_alloc();
	pdf_offset.device_alloc();

	// pdf_orientation is ipart-independent, so we keep it above ipart scope
	CTIC(cudaMLO->timer,"get_orient_priors");
	for (int exp_iclass = sp.iclass_min; exp_iclass <= sp.iclass_max; exp_iclass++)
		for (long int idir = sp.idir_min, iorientclass = (exp_iclass-sp.iclass_min) * sp.nr_dir * sp.nr_psi; idir <=sp.idir_max; idir++)
			for (long int ipsi = sp.ipsi_min; ipsi <= sp.ipsi_max; ipsi++, iorientclass++)
			{
				RFLOAT pdf(0);

				if (baseMLO->do_skip_align || baseMLO->do_skip_rotate)
					pdf = baseMLO->mymodel.pdf_class[exp_iclass];
				else if (baseMLO->mymodel.orientational_prior_mode == NOPRIOR)
					pdf = DIRECT_MULTIDIM_ELEM(baseMLO->mymodel.pdf_direction[exp_iclass], idir);
				else
					pdf = op.directions_prior[idir] * op.psi_prior[ipsi];

				pdf_orientation[iorientclass] = pdf;
				pdf_orientation_mean += pdf;
				pdf_orientation_count ++;
			}


	pdf_orientation_mean /= (RFLOAT) pdf_orientation_count;

	//If mean is non-zero bring all values closer to 1 to improve numerical accuracy
	//This factor is over all classes and is thus removed in the final normalization
	if (pdf_orientation_mean != 0.)
		for (int i = 0; i < pdf_orientation.getSize(); i ++)
			pdf_orientation[i] /= pdf_orientation_mean;

	pdf_orientation.cp_to_device();
	CTOC(cudaMLO->timer,"get_orient_priors");

	if(exp_ipass==0 || baseMLO->adaptive_oversampling!=0)
	{
		op.sum_weight.clear();
		op.sum_weight.resize(sp.nr_particles, (RFLOAT)(sp.nr_particles));
		op.max_weight.clear();
		op.max_weight.resize(sp.nr_particles, (RFLOAT)-1);
	}

	if (exp_ipass==0)
		op.Mcoarse_significant.resizeNoCp(1,1,sp.nr_particles, XSIZE(op.Mweight));

	XFLOAT my_significant_weight;
	op.significant_weight.clear();
	op.significant_weight.resize(sp.nr_particles, 0.);

	// loop over all particles inside this ori_particle
	for (long int ipart = 0; ipart < sp.nr_particles; ipart++)
	{
		long int part_id = baseMLO->mydata.ori_particles[op.my_ori_particle].particles_id[ipart];

		RFLOAT old_offset_z;
		RFLOAT old_offset_x = XX(op.old_offset[ipart]);
		RFLOAT old_offset_y = YY(op.old_offset[ipart]);
		if (cudaMLO->dataIs3D)
			old_offset_z = ZZ(op.old_offset[ipart]);

		if ((baseMLO->iter == 1 && baseMLO->do_firstiter_cc) || baseMLO->do_always_cc)
		{
			if(exp_ipass==0)
			{
				int nr_coarse_weights = (sp.iclass_max-sp.iclass_min+1)*sp.nr_particles * sp.nr_dir * sp.nr_psi * sp.nr_trans;
				PassWeights[ipart].weights.setDevPtr(&Mweight(ipart*nr_coarse_weights));
				PassWeights[ipart].weights.setHstPtr(&Mweight[ipart*nr_coarse_weights]);
				PassWeights[ipart].weights.setSize(nr_coarse_weights);
			}
			PassWeights[ipart].weights.h_do_free=false;

			std::pair<int, XFLOAT> min_pair=getArgMinOnDevice(PassWeights[ipart].weights);
			PassWeights[ipart].weights.cp_to_host();
			DEBUG_HANDLE_ERROR(hipStreamSynchronize(hipStreamPerThread));

			//Set all device-located weights to zero, and only the smallest one to 1.
			DEBUG_HANDLE_ERROR(hipMemsetAsync(~(PassWeights[ipart].weights), 0.f, PassWeights[ipart].weights.getSize()*sizeof(XFLOAT),0));

			XFLOAT unity=1;
			DEBUG_HANDLE_ERROR(hipMemcpyAsync( &(PassWeights[ipart].weights(min_pair.first) ), &unity, sizeof(XFLOAT), hipMemcpyHostToDevice, 0));

			PassWeights[ipart].weights.cp_to_host();
			DEBUG_HANDLE_ERROR(hipStreamSynchronize(hipStreamPerThread));

			my_significant_weight = 0.999;
			DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_NR_SIGN) = (RFLOAT) 1.;
			if (exp_ipass==0) // TODO better memset, 0 => false , 1 => true
				for (int ihidden = 0; ihidden < XSIZE(op.Mcoarse_significant); ihidden++)
					if (DIRECT_A2D_ELEM(op.Mweight, ipart, ihidden) >= my_significant_weight)
						DIRECT_A2D_ELEM(op.Mcoarse_significant, ipart, ihidden) = true;
					else
						DIRECT_A2D_ELEM(op.Mcoarse_significant, ipart, ihidden) = false;
			else
			{
				std::pair<int, XFLOAT> max_pair = getArgMaxOnDevice(PassWeights[ipart].weights);
				op.max_index[ipart].fineIdx = PassWeights[ipart].ihidden_overs[max_pair.first];
				op.max_weight[ipart] = max_pair.second;
			}

		}
		else
		{


			long int sumRedSize=0;
			for (int exp_iclass = sp.iclass_min; exp_iclass <= sp.iclass_max; exp_iclass++)
				sumRedSize+= (exp_ipass==0) ? ceilf((float)(sp.nr_dir*sp.nr_psi)/(float)SUMW_BLOCK_SIZE) : ceil((float)FPCMasks[ipart][exp_iclass].jobNum / (float)SUMW_BLOCK_SIZE);

			// loop through making translational priors for all classes this ipart - then copy all at once - then loop through kernel calls ( TODO: group kernel calls into one big kernel)
			CTIC(cudaMLO->timer,"get_offset_priors");

			double pdf_offset_mean(0);
			std::vector<double> pdf_offset_t(pdf_offset.getSize());
			unsigned pdf_offset_count(0);

			for (int exp_iclass = sp.iclass_min; exp_iclass <= sp.iclass_max; exp_iclass++)
			{
				/*=========================================
						Fetch+generate Translation data
				===========================================*/
				RFLOAT myprior_x, myprior_y, myprior_z;
				if (baseMLO->mymodel.ref_dim == 2)
				{
					myprior_x = XX(baseMLO->mymodel.prior_offset_class[exp_iclass]);
					myprior_y = YY(baseMLO->mymodel.prior_offset_class[exp_iclass]);
				}
				else
				{
					myprior_x = XX(op.prior[ipart]);
					myprior_y = YY(op.prior[ipart]);
					if (cudaMLO->dataIs3D)
						myprior_z = ZZ(op.prior[ipart]);
				}

				for (long int itrans = sp.itrans_min; itrans <= sp.itrans_max; itrans++)
				{
					RFLOAT mypriors_len2 = myprior_x * myprior_x + myprior_y * myprior_y;
					if (cudaMLO->dataIs3D)
						mypriors_len2 += myprior_z * myprior_z;

					// If it is doing helical refinement AND Cartesian vector myprior has a length > 0, transform the vector to its helical coordinates
					if ( (baseMLO->do_helical_refine) && (! baseMLO->ignore_helical_symmetry) && (mypriors_len2 > 0.00001) )
					{
						RFLOAT rot_deg = DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_ROT);
						RFLOAT tilt_deg = DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_TILT);
						RFLOAT psi_deg = DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_PSI);
						transformCartesianAndHelicalCoords(myprior_x, myprior_y, myprior_z, myprior_x, myprior_y, myprior_z, rot_deg, tilt_deg, psi_deg, (cudaMLO->dataIs3D) ? (3) : (2), CART_TO_HELICAL_COORDS);
					}
					// (For helical refinement) Now offset, old_offset, sampling.translations and myprior are all in helical coordinates

					// To speed things up, only calculate pdf_offset at the coarse sampling.
					// That should not matter much, and that way one does not need to calculate all the OversampledTranslations
					double pdf(0);
					RFLOAT offset_x = old_offset_x + baseMLO->sampling.translations_x[itrans];
					RFLOAT offset_y = old_offset_y + baseMLO->sampling.translations_y[itrans];
					double tdiff2 = 0.;

					if ( (! baseMLO->do_helical_refine) || (baseMLO->ignore_helical_symmetry) || (cudaMLO->dataIs3D) )
						tdiff2 += (offset_x - myprior_x) * (offset_x - myprior_x);
					tdiff2 += (offset_y - myprior_y) * (offset_y - myprior_y);
					if (cudaMLO->dataIs3D)
					{
						RFLOAT offset_z = old_offset_z + baseMLO->sampling.translations_z[itrans];
						if ( (! baseMLO->do_helical_refine) || (baseMLO->ignore_helical_symmetry) )
							tdiff2 += (offset_z - myprior_z) * (offset_z - myprior_z);
					}

					// P(offset|sigma2_offset)
					// This is the probability of the offset, given the model offset and variance.
					if (baseMLO->mymodel.sigma2_offset < 0.0001)
						pdf = ( tdiff2 > 0.) ? 0. : 1.;
					else
						pdf = exp ( tdiff2 / (-2. * baseMLO->mymodel.sigma2_offset) ) / ( 2. * PI * baseMLO->mymodel.sigma2_offset );

					pdf_offset_t[(exp_iclass-sp.iclass_min)*sp.nr_trans + itrans] = pdf;
					pdf_offset_mean += pdf;
					pdf_offset_count ++;
				}
			}

			pdf_offset_mean /= (double) pdf_offset_count;

			//If mean is non-zero bring all values closer to 1 to improve numerical accuracy
			//This factor is over all classes and is thus removed in the final normalization
			if (pdf_offset_mean != 0.)
				for (int i = 0; i < pdf_offset.getSize(); i ++)
					pdf_offset[i] = pdf_offset_t[i] /  pdf_offset_mean;

			pdf_offset.cp_to_device();
			CTOC(cudaMLO->timer,"get_offset_priors");
			CTIC(cudaMLO->timer,"sumweight1");

			long int block_num;

			//Make sure most significant value is at least within single precision limit and some slack to distinguish peaks after prior multiplication
			XFLOAT local_norm = (XFLOAT)op.avg_diff2[ipart];
			if (local_norm - op.min_diff2[ipart] > 50)
				local_norm = op.min_diff2[ipart] + 50;

			if(exp_ipass==0)
			{
				CudaGlobalPtr<weights_t> weights(Mweight.getAllocator());
				weights.setSize(Mweight.getSize());

				if (sizeof(weights_t) == sizeof(XFLOAT))
				{
					weights.setHstPtr((weights_t*) Mweight.h_ptr);
					weights.setDevPtr((weights_t*) Mweight.d_ptr);
					weights.setAllocator(Mweight.getAllocator());
				}
				else
				{
					weights.device_alloc();
					block_num = ceilf((float)Mweight.getSize()/(float)BLOCK_SIZE);
					cuda_kernel_cast<XFLOAT,weights_t><<<block_num,BLOCK_SIZE,0>>>
							(~Mweight,~weights,Mweight.getSize());
				}

				CudaGlobalPtr<weights_t>  ipartMweight(
						weights,
						ipart * op.Mweight.xdim + sp.nr_dir * sp.nr_psi * sp.nr_trans * sp.iclass_min,
						(sp.iclass_max-sp.iclass_min+1) * sp.nr_dir * sp.nr_psi * sp.nr_trans);

				block_num = ceilf((float)(sp.nr_dir*sp.nr_psi)/(float)SUMW_BLOCK_SIZE);
				dim3 block_dim(block_num,sp.iclass_max-sp.iclass_min+1);

				if (failsafeMode) //Prevent zero prior products in fail-safe mode
				{
					cuda_kernel_exponentiate_weights_coarse<true,weights_t>
					<<<block_dim,SUMW_BLOCK_SIZE,0>>>(
							~pdf_orientation,
							~pdf_offset,
							~ipartMweight,
							local_norm,
							(XFLOAT)op.min_diff2[ipart],
							sp.nr_dir*sp.nr_psi,
							sp.nr_trans);
				}
				else
				{
					cuda_kernel_exponentiate_weights_coarse<false,weights_t>
					<<<block_dim,SUMW_BLOCK_SIZE,0>>>(
							~pdf_orientation,
							~pdf_offset,
							~ipartMweight,
							local_norm,
							(XFLOAT)op.min_diff2[ipart],
							sp.nr_dir*sp.nr_psi,
							sp.nr_trans);
				}

				CTIC(cudaMLO->timer,"sort");
				DEBUG_HANDLE_ERROR(hipStreamSynchronize(hipStreamPerThread));

				long ipart_length = (sp.iclass_max-sp.iclass_min+1) * sp.nr_dir * sp.nr_psi * sp.nr_trans;

				if (ipart_length > 1)
				{
					//Wrap the current ipart data in a new pointer
					CudaGlobalPtr<weights_t> unsorted_ipart(weights,
							ipart * op.Mweight.xdim + sp.nr_dir * sp.nr_psi * sp.nr_trans * sp.iclass_min,
							ipart_length);

					CudaGlobalPtr<weights_t> filtered(unsorted_ipart.getSize(), cudaMLO->devBundle->allocator);

					CUSTOM_ALLOCATOR_REGION_NAME("CASDTW_SORTSUM");

					filtered.device_alloc();

					MoreThanCubOpt<weights_t> moreThanOpt(0.);
					size_t filteredSize = filterOnDevice(unsorted_ipart, filtered, moreThanOpt);

					if (filteredSize == 0)
					{
						if (failsafeMode) //Only print error if not managed to recover through fail-safe mode
						{
							std::cerr << std::endl;
							std::cerr << " fn_img= " << sp.current_img << std::endl;
							std::cerr << " ipart= " << ipart << " adaptive_fraction= " << baseMLO->adaptive_fraction << std::endl;
							std::cerr << " min_diff2= " << op.min_diff2[ipart] << std::endl;

							pdf_orientation.dump_device_to_file("error_dump_pdf_orientation");
							pdf_offset.dump_device_to_file("error_dump_pdf_offset");
							unsorted_ipart.dump_device_to_file("error_dump_filtered");

							std::cerr << "Dumped data: error_dump_pdf_orientation, error_dump_pdf_orientation and error_dump_unsorted." << std::endl;
						}

						CRITICAL(ERRFILTEREDZERO); // "filteredSize == 0"
					}
					filtered.setSize(filteredSize);

					CudaGlobalPtr<weights_t> sorted(filteredSize, cudaMLO->devBundle->allocator);
					CudaGlobalPtr<weights_t> cumulative_sum(filteredSize, cudaMLO->devBundle->allocator);
					sorted.device_alloc();
					cumulative_sum.device_alloc();

					sortOnDevice(filtered, sorted);
					scanOnDevice(sorted, cumulative_sum);

					CTOC(cudaMLO->timer,"sort");

					op.sum_weight[ipart] = cumulative_sum.getDeviceAt(cumulative_sum.getSize() - 1);

					long int my_nr_significant_coarse_samples;
					size_t thresholdIdx(0);

					int grid_size = ceil((float)(cumulative_sum.getSize()-1)/(float)FIND_IN_CUMULATIVE_BLOCK_SIZE);
					if(grid_size > 0)
					{
						CudaGlobalPtr<size_t >  idx(1, cumulative_sum.getStream(), cumulative_sum.getAllocator());
						idx[0] = 0;
						idx.put_on_device();
						cuda_kernel_find_threshold_idx_in_cumulative<weights_t>
						<<< grid_size, FIND_IN_CUMULATIVE_BLOCK_SIZE, 0, cumulative_sum.getStream() >>>(
								~cumulative_sum,
								(1 - baseMLO->adaptive_fraction) * op.sum_weight[ipart],
								cumulative_sum.getSize()-1,
								~idx);
						idx.cp_to_host();
						DEBUG_HANDLE_ERROR(hipStreamSynchronize(cumulative_sum.getStream()));

						thresholdIdx = idx[0];
					}


					my_nr_significant_coarse_samples = filteredSize - thresholdIdx;

					if (my_nr_significant_coarse_samples == 0)
					{
						if (failsafeMode) //Only print error if not managed to recover through fail-safe mode
						{
							std::cerr << std::endl;
							std::cerr << " fn_img= " << sp.current_img << std::endl;
							std::cerr << " ipart= " << ipart << " adaptive_fraction= " << baseMLO->adaptive_fraction << std::endl;
							std::cerr << " threshold= " << (1 - baseMLO->adaptive_fraction) * op.sum_weight[ipart] << " thresholdIdx= " << thresholdIdx << std::endl;
							std::cerr << " op.sum_weight[ipart]= " << op.sum_weight[ipart] << std::endl;
							std::cerr << " min_diff2= " << op.min_diff2[ipart] << std::endl;

							unsorted_ipart.dump_device_to_file("error_dump_unsorted");
							filtered.dump_device_to_file("error_dump_filtered");
							sorted.dump_device_to_file("error_dump_sorted");
							cumulative_sum.dump_device_to_file("error_dump_cumulative_sum");

							std::cerr << "Written error_dump_unsorted, error_dump_filtered, error_dump_sorted, and error_dump_cumulative_sum." << std::endl;
						}

						CRITICAL(ERRNOSIGNIFS); // "my_nr_significant_coarse_samples == 0"
					}

					if (baseMLO->maximum_significants != 0 &&
							my_nr_significant_coarse_samples > baseMLO->maximum_significants)
					{
						my_nr_significant_coarse_samples = baseMLO->maximum_significants;
						thresholdIdx = filteredSize - my_nr_significant_coarse_samples;
					}

					weights_t significant_weight = sorted.getDeviceAt(thresholdIdx);

					CTIC(cudaMLO->timer,"getArgMaxOnDevice");
					std::pair<int, weights_t> max_pair = getArgMaxOnDevice(unsorted_ipart);
					CTOC(cudaMLO->timer,"getArgMaxOnDevice");
					op.max_index[ipart].coarseIdx = max_pair.first;
					op.max_weight[ipart] = max_pair.second;

					// Store nr_significant_coarse_samples for this particle
					DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_NR_SIGN) = (RFLOAT) my_nr_significant_coarse_samples;

					CudaGlobalPtr<bool> Mcoarse_significant(
							&op.Mcoarse_significant.data[ipart * op.Mweight.xdim + sp.nr_dir * sp.nr_psi * sp.nr_trans * sp.iclass_min],
							(sp.iclass_max-sp.iclass_min+1) * sp.nr_dir * sp.nr_psi * sp.nr_trans,
							cudaMLO->devBundle->allocator);

					CUSTOM_ALLOCATOR_REGION_NAME("CASDTW_SIG");
					Mcoarse_significant.device_alloc();

					DEBUG_HANDLE_ERROR(hipStreamSynchronize(hipStreamPerThread));
					arrayOverThreshold<weights_t>(unsorted_ipart, Mcoarse_significant, significant_weight);
					Mcoarse_significant.cp_to_host();
					DEBUG_HANDLE_ERROR(hipStreamSynchronize(hipStreamPerThread));
				}
				else if (ipart_length == 1)
				{
					op.Mcoarse_significant.data[ipart * op.Mweight.xdim + sp.nr_dir * sp.nr_psi * sp.nr_trans * sp.iclass_min] = 1;
				}
				else
					CRITICAL(ERRNEGLENGTH);
			}
			else
			{

				for (int exp_iclass = sp.iclass_min; exp_iclass <= sp.iclass_max; exp_iclass++)
					DEBUG_HANDLE_ERROR(hipStreamSynchronize(cudaMLO->classStreams[exp_iclass]));
				DEBUG_HANDLE_ERROR(hipStreamSynchronize(hipStreamPerThread));

				for (int exp_iclass = sp.iclass_min; exp_iclass <= sp.iclass_max; exp_iclass++) // TODO could use classStreams
				{
					if ((baseMLO->mymodel.pdf_class[exp_iclass] > 0.) && (FPCMasks[ipart][exp_iclass].weightNum > 0) )
					{
						// Use the constructed mask to build a partial (class-specific) input
						// (until now, PassWeights has been an empty placeholder. We now create class-paritals pointing at it, and start to fill it with stuff)
						IndexedDataArray thisClassPassWeights(PassWeights[ipart],FPCMasks[ipart][exp_iclass], cudaMLO->devBundle->allocator);
						CudaGlobalPtr<XFLOAT>  pdf_orientation_class(&(pdf_orientation[(exp_iclass-sp.iclass_min)*sp.nr_dir*sp.nr_psi]), &( pdf_orientation((exp_iclass-sp.iclass_min)*sp.nr_dir*sp.nr_psi) ), sp.nr_dir*sp.nr_psi);
						CudaGlobalPtr<XFLOAT>  pdf_offset_class(&(pdf_offset[(exp_iclass-sp.iclass_min)*sp.nr_trans]), &( pdf_offset((exp_iclass-sp.iclass_min)*sp.nr_trans) ), sp.nr_trans);

						block_num = ceil((float)FPCMasks[ipart][exp_iclass].jobNum / (float)SUMW_BLOCK_SIZE); //thisClassPassWeights.rot_idx.getSize() / SUM_BLOCK_SIZE;
						dim3 block_dim(block_num);

						cuda_kernel_exponentiate_weights_fine<<<block_dim,SUMW_BLOCK_SIZE,0,cudaMLO->classStreams[exp_iclass]>>>(
								~pdf_orientation_class,
								~pdf_offset_class,
								~thisClassPassWeights.weights,
								(XFLOAT)local_norm,
								sp.nr_oversampled_rot,
								sp.nr_oversampled_trans,
								~thisClassPassWeights.rot_id,
								~thisClassPassWeights.trans_idx,
								~FPCMasks[ipart][exp_iclass].jobOrigin,
								~FPCMasks[ipart][exp_iclass].jobExtent,
								FPCMasks[ipart][exp_iclass].jobNum);
								LAUNCH_PRIVATE_ERROR(hipGetLastError(),cudaMLO->errorStatus);
					}

				}

				for (int exp_iclass = sp.iclass_min; exp_iclass <= sp.iclass_max; exp_iclass++)
					DEBUG_HANDLE_ERROR(hipStreamSynchronize(cudaMLO->classStreams[exp_iclass]));
				DEBUG_HANDLE_ERROR(hipStreamSynchronize(hipStreamPerThread));

				PassWeights[ipart].weights.cp_to_host(); // note that the host-pointer is shared: we're copying to Mweight.


				CTIC(cudaMLO->timer,"sort");
				DEBUG_HANDLE_ERROR(hipStreamSynchronize(hipStreamPerThread));
				size_t weightSize = PassWeights[ipart].weights.getSize();

				CudaGlobalPtr<XFLOAT> sorted(weightSize, cudaMLO->devBundle->allocator);
				CudaGlobalPtr<XFLOAT> cumulative_sum(weightSize, cudaMLO->devBundle->allocator);

				CUSTOM_ALLOCATOR_REGION_NAME("CASDTW_FINE");

				sorted.device_alloc();
				cumulative_sum.device_alloc();

				sortOnDevice(PassWeights[ipart].weights, sorted);
				scanOnDevice(sorted, cumulative_sum);
				CTOC(cudaMLO->timer,"sort");

				if(baseMLO->adaptive_oversampling!=0)
				{
					op.sum_weight[ipart] = cumulative_sum.getDeviceAt(cumulative_sum.getSize() - 1);

					if (op.sum_weight[ipart]==0)
					{
						std::cerr << std::endl;
						std::cerr << " fn_img= " << sp.current_img << std::endl;
						std::cerr << " part_id= " << part_id << std::endl;
						std::cerr << " ipart= " << ipart << std::endl;
						std::cerr << " op.min_diff2[ipart]= " << op.min_diff2[ipart] << std::endl;
						int group_id = baseMLO->mydata.getGroupId(part_id);
						std::cerr << " group_id= " << group_id << std::endl;
						std::cerr << " ml_model.scale_correction[group_id]= " << baseMLO->mymodel.scale_correction[group_id] << std::endl;
						std::cerr << " exp_significant_weight[ipart]= " << op.significant_weight[ipart] << std::endl;
						std::cerr << " exp_max_weight[ipart]= " << op.max_weight[ipart] << std::endl;
						std::cerr << " ml_model.sigma2_noise[group_id]= " << baseMLO->mymodel.sigma2_noise[group_id] << std::endl;
						CRITICAL(ERRSUMWEIGHTZERO); //"op.sum_weight[ipart]==0"
					}

					size_t thresholdIdx = findThresholdIdxInCumulativeSum(cumulative_sum, (1 - baseMLO->adaptive_fraction) * op.sum_weight[ipart]);
					my_significant_weight = sorted.getDeviceAt(thresholdIdx);

					CTIC(cudaMLO->timer,"getArgMaxOnDevice");
					std::pair<int, XFLOAT> max_pair = getArgMaxOnDevice(PassWeights[ipart].weights);
					CTOC(cudaMLO->timer,"getArgMaxOnDevice");
					op.max_index[ipart].fineIdx = PassWeights[ipart].ihidden_overs[max_pair.first];
					op.max_weight[ipart] = max_pair.second;
				}
				else
				{
					my_significant_weight = sorted.getDeviceAt(0);
				}
			}
			CTOC(cudaMLO->timer,"sumweight1");
		}

		op.significant_weight[ipart] = (RFLOAT) my_significant_weight;
	} // end loop ipart

#ifdef TIMING
	if (op.my_ori_particle == baseMLO->exp_my_first_ori_particle)
	{
		if (exp_ipass == 0) baseMLO->timer.toc(baseMLO->TIMING_ESP_WEIGHT1);
		else baseMLO->timer.toc(baseMLO->TIMING_ESP_WEIGHT2);
	}
#endif
}

void storeWeightedSums(OptimisationParamters &op, SamplingParameters &sp,
						MlOptimiser *baseMLO,
						MlOptimiserCuda *cudaMLO,
						std::vector<IndexedDataArray> &FinePassWeights,
						std::vector<ProjectionParams> &ProjectionData,
						std::vector<std::vector<IndexedDataArrayMask> > &FPCMasks,
	 	 	 	 	 	std::vector<cudaStager<unsigned long> > &stagerSWS)
{
#ifdef TIMING
	if (op.my_ori_particle == baseMLO->exp_my_first_ori_particle)
		baseMLO->timer.tic(baseMLO->TIMING_ESP_WSUM);
#endif
	CTIC(cudaMLO->timer,"store_init");

	int ibody(0); //Not supported yet

	// Re-do below because now also want unmasked images AND if (stricht_highres_exp >0.) then may need to resize
	std::vector<MultidimArray<Complex > > dummy;
	baseMLO->precalculateShiftedImagesCtfsAndInvSigma2s(false, op.my_ori_particle, sp.current_image_size, sp.current_oversampling, op.metadata_offset, // inserted SHWS 12112015
			sp.itrans_min, sp.itrans_max, op.Fimgs, op.Fimgs_nomask, op.Fctfs, dummy, dummy,
			op.local_Fctfs, op.local_sqrtXi2, op.local_Minvsigma2s);

	// In doThreadPrecalculateShiftedImagesCtfsAndInvSigma2s() the origin of the op.local_Minvsigma2s was omitted.
	// Set those back here
	for (long int ipart = 0; ipart < sp.nr_particles; ipart++)
	{
		long int part_id = baseMLO->mydata.ori_particles[op.my_ori_particle].particles_id[ipart];
		int group_id = baseMLO->mydata.getGroupId(part_id);
		DIRECT_MULTIDIM_ELEM(op.local_Minvsigma2s[ipart], 0) = 1. / (baseMLO->sigma2_fudge * DIRECT_A1D_ELEM(baseMLO->mymodel.sigma2_noise[group_id], 0));
	}

	// For norm_correction and scale_correction of all particles of this ori_particle
	std::vector<RFLOAT> exp_wsum_norm_correction;
	std::vector<MultidimArray<RFLOAT> > exp_wsum_scale_correction_XA, exp_wsum_scale_correction_AA;
	std::vector<MultidimArray<RFLOAT> > thr_wsum_signal_product_spectra, thr_wsum_reference_power_spectra;
	exp_wsum_norm_correction.resize(sp.nr_particles, 0.);

	// For scale_correction
	if (baseMLO->do_scale_correction)
	{
		MultidimArray<RFLOAT> aux;
		aux.initZeros(baseMLO->mymodel.ori_size/2 + 1);
		exp_wsum_scale_correction_XA.resize(sp.nr_particles, aux);
		exp_wsum_scale_correction_AA.resize(sp.nr_particles, aux);
		thr_wsum_signal_product_spectra.resize(baseMLO->mymodel.nr_groups, aux);
		thr_wsum_reference_power_spectra.resize(baseMLO->mymodel.nr_groups, aux);
	}

	std::vector<RFLOAT> oversampled_translations_x, oversampled_translations_y, oversampled_translations_z;
	bool have_warned_small_scale = false;

	// Make local copies of weighted sums (except BPrefs, which are too big)
	// so that there are not too many mutex locks below
	std::vector<MultidimArray<RFLOAT> > thr_wsum_sigma2_noise, thr_wsum_pdf_direction;
	std::vector<RFLOAT> thr_wsum_norm_correction, thr_sumw_group, thr_wsum_pdf_class, thr_wsum_prior_offsetx_class, thr_wsum_prior_offsety_class;
	RFLOAT thr_wsum_sigma2_offset;
	MultidimArray<RFLOAT> thr_metadata, zeroArray;
	// Wsum_sigma_noise2 is a 1D-spectrum for each group
	zeroArray.initZeros(baseMLO->mymodel.ori_size/2 + 1);
	thr_wsum_sigma2_noise.resize(baseMLO->mymodel.nr_groups, zeroArray);
	// wsum_pdf_direction is a 1D-array (of length sampling.NrDirections()) for each class
	zeroArray.initZeros(baseMLO->sampling.NrDirections());
	thr_wsum_pdf_direction.resize(baseMLO->mymodel.nr_classes, zeroArray);
	// sumw_group is a RFLOAT for each group
	thr_sumw_group.resize(baseMLO->mymodel.nr_groups, 0.);
	// wsum_pdf_class is a RFLOAT for each class
	thr_wsum_pdf_class.resize(baseMLO->mymodel.nr_classes, 0.);
	if (baseMLO->mymodel.ref_dim == 2)
	{
		thr_wsum_prior_offsetx_class.resize(baseMLO->mymodel.nr_classes, 0.);
		thr_wsum_prior_offsety_class.resize(baseMLO->mymodel.nr_classes, 0.);
	}
	// wsum_sigma2_offset is just a RFLOAT
	thr_wsum_sigma2_offset = 0.;
	unsigned image_size = op.Fimgs[0].nzyxdim;

	CTOC(cudaMLO->timer,"store_init");

	/*=======================================================================================
	                           COLLECT 2 AND SET METADATA
	=======================================================================================*/

	CTIC(cudaMLO->timer,"collect_data_2");
	int nr_transes = sp.nr_trans*sp.nr_oversampled_trans;
	int nr_fake_classes = (sp.iclass_max-sp.iclass_min+1);
	int oversamples = sp.nr_oversampled_trans * sp.nr_oversampled_rot;
	std::vector<long int> block_nums(sp.nr_particles*nr_fake_classes);

	for (long int ipart = 0; ipart < sp.nr_particles; ipart++)
	{
		// Allocate space for all classes, so that we can pre-calculate data for all classes, copy in one operation, call kenrels on all classes, and copy back in one operation
		CudaGlobalPtr<XFLOAT>          oo_otrans_x(nr_fake_classes*nr_transes, cudaMLO->devBundle->allocator); // old_offset_oversampled_trans_x
		CudaGlobalPtr<XFLOAT>          oo_otrans_y(nr_fake_classes*nr_transes, cudaMLO->devBundle->allocator);
		CudaGlobalPtr<XFLOAT>          oo_otrans_z(nr_fake_classes*nr_transes, cudaMLO->devBundle->allocator);
		CudaGlobalPtr<XFLOAT> myp_oo_otrans_x2y2z2(nr_fake_classes*nr_transes, cudaMLO->devBundle->allocator); // my_prior_old_offs....x^2*y^2*z^2

		myp_oo_otrans_x2y2z2.device_alloc();

		int sumBlockNum =0;
		long int part_id = baseMLO->mydata.ori_particles[op.my_ori_particle].particles_id[ipart];
		int group_id = baseMLO->mydata.getGroupId(part_id);
		CTIC(cudaMLO->timer,"collect_data_2_pre_kernel");
		for (int exp_iclass = sp.iclass_min; exp_iclass <= sp.iclass_max; exp_iclass++)
		{
			int fake_class = exp_iclass-sp.iclass_min; // if we only have the third class to do, the third class will be the "first" we do, i.e. the "fake" first.
			if ((baseMLO->mymodel.pdf_class[exp_iclass] == 0.) || (ProjectionData[ipart].class_entries[exp_iclass] == 0) )
				continue;

			// Use the constructed mask to construct a partial class-specific input
			IndexedDataArray thisClassFinePassWeights(FinePassWeights[ipart],FPCMasks[ipart][exp_iclass], cudaMLO->devBundle->allocator);

			// Re-define the job-partition of the indexedArray of weights so that the collect-kernel can work with it.
			block_nums[nr_fake_classes*ipart + fake_class] = makeJobsForCollect(thisClassFinePassWeights, FPCMasks[ipart][exp_iclass], ProjectionData[ipart].orientation_num[exp_iclass]);

			stagerSWS[ipart].stage(FPCMasks[ipart][exp_iclass].jobOrigin);
			stagerSWS[ipart].stage(FPCMasks[ipart][exp_iclass].jobExtent);

			sumBlockNum+=block_nums[nr_fake_classes*ipart + fake_class];

			RFLOAT myprior_x, myprior_y, myprior_z;
			RFLOAT old_offset_x = XX(op.old_offset[ipart]);
			RFLOAT old_offset_y = YY(op.old_offset[ipart]);
			RFLOAT old_offset_z;

			if (baseMLO->mymodel.ref_dim == 2)
			{
				myprior_x = XX(baseMLO->mymodel.prior_offset_class[exp_iclass]);
				myprior_y = YY(baseMLO->mymodel.prior_offset_class[exp_iclass]);
			}
			else
			{
				myprior_x = XX(op.prior[ipart]);
				myprior_y = YY(op.prior[ipart]);
				if (cudaMLO->dataIs3D)
				{
					myprior_z = ZZ(op.prior[ipart]);
					old_offset_z = ZZ(op.old_offset[ipart]);
				}
			}

			/*======================================================
								COLLECT 2
			======================================================*/

			//Pregenerate oversampled translation objects for kernel-call
			for (long int itrans = 0, iitrans = 0; itrans < sp.nr_trans; itrans++)
			{
				baseMLO->sampling.getTranslations(itrans, baseMLO->adaptive_oversampling,
						oversampled_translations_x, oversampled_translations_y, oversampled_translations_z,
						(baseMLO->do_helical_refine) && (! baseMLO->ignore_helical_symmetry), baseMLO->helical_rise_initial / baseMLO->mymodel.pixel_size, baseMLO->helical_twist_initial);
				for (long int iover_trans = 0; iover_trans < sp.nr_oversampled_trans; iover_trans++, iitrans++)
				{
					oo_otrans_x[fake_class*nr_transes+iitrans] = old_offset_x + oversampled_translations_x[iover_trans];
					oo_otrans_y[fake_class*nr_transes+iitrans] = old_offset_y + oversampled_translations_y[iover_trans];
					if (cudaMLO->dataIs3D)
						oo_otrans_z[fake_class*nr_transes+iitrans] = old_offset_z + oversampled_translations_z[iover_trans];

					// Calculate the vector length of myprior
					RFLOAT mypriors_len2 = myprior_x * myprior_x + myprior_y * myprior_y;
					if (cudaMLO->dataIs3D)
						mypriors_len2 += myprior_z * myprior_z;

					// If it is doing helical refinement AND Cartesian vector myprior has a length > 0, transform the vector to its helical coordinates
					if ( (baseMLO->do_helical_refine) && (! baseMLO->ignore_helical_symmetry) && (mypriors_len2 > 0.00001) )
					{
						RFLOAT rot_deg = DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_ROT);
						RFLOAT tilt_deg = DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_TILT);
						RFLOAT psi_deg = DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_PSI);
						transformCartesianAndHelicalCoords(myprior_x, myprior_y, myprior_z, myprior_x, myprior_y, myprior_z, rot_deg, tilt_deg, psi_deg, (cudaMLO->dataIs3D) ? (3) : (2), CART_TO_HELICAL_COORDS);
					}

					// TODO: Feb20,2017 - Shaoda does not understand what you are doing here ... ???
					// Please check whether the following is compatible with 3D reconstructions of 2D helical segments AND 3D helical subtomograms ???
					// Preliminary tests show that the code from Shaoda gave worse reconstructions of VipA/VipB (EMPIAR-10019)
					// While TMV (EMPIAR-10020) and BtubAB subtomo results are not affected
					// ========= OLD ===========
					if ( (! baseMLO->do_helical_refine) || (baseMLO->ignore_helical_symmetry) )
						RFLOAT diffx = myprior_x - oo_otrans_x[fake_class*nr_transes+iitrans];
					RFLOAT diffx = myprior_x - oo_otrans_x[fake_class*nr_transes+iitrans];
					RFLOAT diffy = myprior_y - oo_otrans_y[fake_class*nr_transes+iitrans];
					RFLOAT diffz = 0;
					if (cudaMLO->dataIs3D)
						diffz = myprior_z - (old_offset_z + oversampled_translations_z[iover_trans]);
					// ======= SHAODA ==========
					//RFLOAT diffx = 0.;
					//if ( (! baseMLO->do_helical_refine) || (baseMLO->ignore_helical_symmetry) || (cudaMLO->dataIs3D) )
					//	diffx = myprior_x - oo_otrans_x[fake_class*nr_transes+iitrans];
					//RFLOAT diffy = myprior_y - oo_otrans_y[fake_class*nr_transes+iitrans];
					//RFLOAT diffz = 0;
					//if (cudaMLO->dataIs3D)
					//{
					//	if ( (! baseMLO->do_helical_refine) || (baseMLO->ignore_helical_symmetry) )
					//		diffz = myprior_z - (old_offset_z + oversampled_translations_z[iover_trans]);
					//}

					myp_oo_otrans_x2y2z2[fake_class*nr_transes+iitrans] = diffx*diffx + diffy*diffy + diffz*diffz;
				}
			}
		}

		stagerSWS[ipart].cp_to_device();
		oo_otrans_x.put_on_device();
		oo_otrans_y.put_on_device();
		oo_otrans_z.put_on_device();

		myp_oo_otrans_x2y2z2.cp_to_device();
		DEBUG_HANDLE_ERROR(hipStreamSynchronize(hipStreamPerThread));

		CudaGlobalPtr<XFLOAT>                      p_weights(sumBlockNum, cudaMLO->devBundle->allocator);
		CudaGlobalPtr<XFLOAT> p_thr_wsum_prior_offsetx_class(sumBlockNum, cudaMLO->devBundle->allocator);
		CudaGlobalPtr<XFLOAT> p_thr_wsum_prior_offsety_class(sumBlockNum, cudaMLO->devBundle->allocator);
		CudaGlobalPtr<XFLOAT> p_thr_wsum_prior_offsetz_class(sumBlockNum, cudaMLO->devBundle->allocator);
		CudaGlobalPtr<XFLOAT>       p_thr_wsum_sigma2_offset(sumBlockNum, cudaMLO->devBundle->allocator);
		p_weights.device_alloc();
		p_thr_wsum_prior_offsetx_class.device_alloc();
		p_thr_wsum_prior_offsety_class.device_alloc();

		if (cudaMLO->dataIs3D)
			p_thr_wsum_prior_offsetz_class.device_alloc();
		else
			p_thr_wsum_prior_offsetz_class.d_ptr  = p_thr_wsum_prior_offsety_class.d_ptr;

		p_thr_wsum_sigma2_offset.device_alloc();
		CTOC(cudaMLO->timer,"collect_data_2_pre_kernel");
		int partial_pos=0;


		for (int exp_iclass = sp.iclass_min; exp_iclass <= sp.iclass_max; exp_iclass++)
		{
			int fake_class = exp_iclass-sp.iclass_min; // if we only have the third class to do, the third class will be the "first" we do, i.e. the "fake" first.
			if ((baseMLO->mymodel.pdf_class[exp_iclass] == 0.) || (ProjectionData[ipart].class_entries[exp_iclass] == 0) )
				continue;

			// Use the constructed mask to construct a partial class-specific input
			IndexedDataArray thisClassFinePassWeights(FinePassWeights[ipart],FPCMasks[ipart][exp_iclass], cudaMLO->devBundle->allocator);


			int cpos=fake_class*nr_transes;
			int block_num = block_nums[nr_fake_classes*ipart + fake_class];
			dim3 grid_dim_collect2 = block_num;

			runCollect2jobs(grid_dim_collect2,
						&(oo_otrans_x(cpos) ),          // otrans-size -> make const
						&(oo_otrans_y(cpos) ),          // otrans-size -> make const
						&(oo_otrans_z(cpos) ),          // otrans-size -> make const
						&(myp_oo_otrans_x2y2z2(cpos) ), // otrans-size -> make const
						~thisClassFinePassWeights.weights,
						(XFLOAT)op.significant_weight[ipart],
						(XFLOAT)op.sum_weight[ipart],
						sp.nr_trans,
						sp.nr_oversampled_trans,
						sp.nr_oversampled_rot,
						oversamples,
						(baseMLO->do_skip_align || baseMLO->do_skip_rotate ),
						&p_weights(partial_pos),
						&p_thr_wsum_prior_offsetx_class(partial_pos),
						&p_thr_wsum_prior_offsety_class(partial_pos),
						&p_thr_wsum_prior_offsetz_class(partial_pos),
						&p_thr_wsum_sigma2_offset(partial_pos),
						~thisClassFinePassWeights.rot_idx,
						~thisClassFinePassWeights.trans_idx,
						~FPCMasks[ipart][exp_iclass].jobOrigin,
						~FPCMasks[ipart][exp_iclass].jobExtent,
						cudaMLO->dataIs3D);
			LAUNCH_PRIVATE_ERROR(hipGetLastError(),cudaMLO->errorStatus);

			partial_pos+=block_num;
		}

		CTIC(cudaMLO->timer,"collect_data_2_post_kernel");
		p_weights.cp_to_host();
		p_thr_wsum_sigma2_offset.cp_to_host();
		p_thr_wsum_prior_offsetx_class.cp_to_host();
		p_thr_wsum_prior_offsety_class.cp_to_host();
		if (cudaMLO->dataIs3D)
			p_thr_wsum_prior_offsetz_class.cp_to_host();

		DEBUG_HANDLE_ERROR(hipStreamSynchronize(hipStreamPerThread));
		int iorient = 0;
		partial_pos=0;
		for (int exp_iclass = sp.iclass_min; exp_iclass <= sp.iclass_max; exp_iclass++)
		{
			int fake_class = exp_iclass-sp.iclass_min; // if we only have the third class to do, the third class will be the "first" we do, i.e. the "fake" first.
			if ((baseMLO->mymodel.pdf_class[exp_iclass] == 0.) || (ProjectionData[ipart].class_entries[exp_iclass] == 0) )
				continue;
			int block_num = block_nums[nr_fake_classes*ipart + fake_class];

			for (long int n = partial_pos; n < partial_pos+block_num; n++)
			{
				iorient= FinePassWeights[ipart].rot_id[FPCMasks[ipart][exp_iclass].jobOrigin[n-partial_pos]+FPCMasks[ipart][exp_iclass].firstPos];

				long int mydir, idir=floor(iorient/sp.nr_psi);
				if (baseMLO->mymodel.orientational_prior_mode == NOPRIOR)
					mydir = idir;
				else
					mydir = op.pointer_dir_nonzeroprior[idir];

				// store partials according to indices of the relevant dimension
				DIRECT_MULTIDIM_ELEM(thr_wsum_pdf_direction[exp_iclass], mydir) += p_weights[n];
				thr_sumw_group[group_id]                 						+= p_weights[n];
				thr_wsum_pdf_class[exp_iclass]           						+= p_weights[n];
				thr_wsum_sigma2_offset                   						+= p_thr_wsum_sigma2_offset[n];

				if (baseMLO->mymodel.ref_dim == 2)
				{
					thr_wsum_prior_offsetx_class[exp_iclass] += p_thr_wsum_prior_offsetx_class[n];
					thr_wsum_prior_offsety_class[exp_iclass] += p_thr_wsum_prior_offsety_class[n];
				}
			}
			partial_pos+=block_num;
		} // end loop iclass
		CTOC(cudaMLO->timer,"collect_data_2_post_kernel");
	} // end loop ipart

	/*======================================================
	                     SET METADATA
	======================================================*/

	std::vector< RFLOAT> oversampled_rot, oversampled_tilt, oversampled_psi;
	for (long int ipart = 0; ipart < sp.nr_particles; ipart++)
	{
		CTIC(cudaMLO->timer,"setMetadata");

//		CTIC(cudaMLO->timer,"getArgMaxOnDevice");
//		std::pair<int, XFLOAT> max_pair = getArgMaxOnDevice(FinePassWeights[ipart].weights);
//		CTOC(cudaMLO->timer,"getArgMaxOnDevice");
//		op.max_index.fineIdx = FinePassWeights[ipart].ihidden_overs[max_pair.first];
//		op.max_weight[ipart] = max_pair.second;


		//std::cerr << "max val = " << op.max_weight[ipart] << std::endl;
		//std::cerr << "max index = " << max_index.fineIdx << std::endl;

		if(baseMLO->adaptive_oversampling!=0)
			op.max_index[ipart].fineIndexToFineIndices(sp); // set partial indices corresponding to the found max_index, to be used below
		else
			op.max_index[ipart].coarseIndexToCoarseIndices(sp);

		baseMLO->sampling.getTranslations(op.max_index[ipart].itrans, baseMLO->adaptive_oversampling,
				oversampled_translations_x, oversampled_translations_y, oversampled_translations_z,
				(baseMLO->do_helical_refine) && (! baseMLO->ignore_helical_symmetry), baseMLO->helical_rise_initial / baseMLO->mymodel.pixel_size, baseMLO->helical_twist_initial);

		//TODO We already have rot, tilt and psi don't calculated them again
		if(baseMLO->do_skip_align || baseMLO->do_skip_rotate)
			   baseMLO->sampling.getOrientations(sp.idir_min, sp.ipsi_min, baseMLO->adaptive_oversampling, oversampled_rot, oversampled_tilt, oversampled_psi,
					   op.pointer_dir_nonzeroprior, op.directions_prior, op.pointer_psi_nonzeroprior, op.psi_prior);
		else
			   baseMLO->sampling.getOrientations(op.max_index[ipart].idir, op.max_index[ipart].ipsi, baseMLO->adaptive_oversampling, oversampled_rot, oversampled_tilt, oversampled_psi,
					op.pointer_dir_nonzeroprior, op.directions_prior, op.pointer_psi_nonzeroprior, op.psi_prior);

		baseMLO->sampling.getOrientations(op.max_index[ipart].idir, op.max_index[ipart].ipsi, baseMLO->adaptive_oversampling, oversampled_rot, oversampled_tilt, oversampled_psi,
				op.pointer_dir_nonzeroprior, op.directions_prior, op.pointer_psi_nonzeroprior, op.psi_prior);

		RFLOAT rot = oversampled_rot[op.max_index[ipart].ioverrot];
		RFLOAT tilt = oversampled_tilt[op.max_index[ipart].ioverrot];
		RFLOAT psi = oversampled_psi[op.max_index[ipart].ioverrot];

		int icol_rot  = (baseMLO->mymodel.nr_bodies == 1) ? METADATA_ROT  : 0 + METADATA_LINE_LENGTH_BEFORE_BODIES + (ibody) * METADATA_NR_BODY_PARAMS;
		int icol_tilt = (baseMLO->mymodel.nr_bodies == 1) ? METADATA_TILT : 1 + METADATA_LINE_LENGTH_BEFORE_BODIES + (ibody) * METADATA_NR_BODY_PARAMS;
		int icol_psi  = (baseMLO->mymodel.nr_bodies == 1) ? METADATA_PSI  : 2 + METADATA_LINE_LENGTH_BEFORE_BODIES + (ibody) * METADATA_NR_BODY_PARAMS;
		int icol_xoff = (baseMLO->mymodel.nr_bodies == 1) ? METADATA_XOFF : 3 + METADATA_LINE_LENGTH_BEFORE_BODIES + (ibody) * METADATA_NR_BODY_PARAMS;
		int icol_yoff = (baseMLO->mymodel.nr_bodies == 1) ? METADATA_YOFF : 4 + METADATA_LINE_LENGTH_BEFORE_BODIES + (ibody) * METADATA_NR_BODY_PARAMS;
		int icol_zoff = (baseMLO->mymodel.nr_bodies == 1) ? METADATA_ZOFF : 5 + METADATA_LINE_LENGTH_BEFORE_BODIES + (ibody) * METADATA_NR_BODY_PARAMS;

		RFLOAT old_rot = DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, icol_rot);
		DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, icol_rot) = rot;
		RFLOAT old_tilt = DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, icol_tilt);
		DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, icol_tilt) = tilt;
		RFLOAT old_psi = DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, icol_psi);
		DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, icol_psi) = psi;

		Matrix1D<RFLOAT> shifts(2);
		//21may2015
		if (baseMLO->mymodel.nr_bodies == 1)
		{
			// include old_offsets for normal refinement (i.e. non multi-body)
			XX(shifts) = XX(op.old_offset[ipart]) + oversampled_translations_x[op.max_index[ipart].iovertrans];
			YY(shifts) = YY(op.old_offset[ipart]) + oversampled_translations_y[op.max_index[ipart].iovertrans];
		}
		else
		{
			// For multi-body refinements, only store 'residual' translations
			XX(shifts) = oversampled_translations_x[op.max_index[ipart].iovertrans];
			YY(shifts) = oversampled_translations_y[op.max_index[ipart].iovertrans];
		}
		if (cudaMLO->dataIs3D)
		{
			shifts.resize(3);
			if (baseMLO->mymodel.nr_bodies == 1)
				ZZ(shifts) = ZZ(op.old_offset[ipart]) + oversampled_translations_z[op.max_index[ipart].iovertrans];
			else
				ZZ(shifts) = oversampled_translations_z[op.max_index[ipart].iovertrans];
		}

		// Use oldpsi-angle to rotate back the XX(exp_old_offset[ipart]) + oversampled_translations_x[iover_trans] and
		if ( (baseMLO->do_helical_refine) && (! baseMLO->ignore_helical_symmetry) )
			transformCartesianAndHelicalCoords(shifts, shifts, old_rot, old_tilt, old_psi, HELICAL_TO_CART_COORDS);

		DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, icol_xoff) = XX(shifts);
		DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, icol_yoff) = YY(shifts);
		if (cudaMLO->dataIs3D)
			DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, icol_zoff) = ZZ(shifts);

		if (ibody == 0)
		{
			DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_CLASS) = (RFLOAT)op.max_index[ipart].iclass + 1;
			RFLOAT pmax = op.max_weight[ipart]/op.sum_weight[ipart];
			if(pmax>1) //maximum normalised probability weight is (unreasonably) larger than unity
				CRITICAL("Relion is finding a normalised probability greater than 1");
			DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_PMAX) = pmax;
		}
		CTOC(cudaMLO->timer,"setMetadata");
	}
	CTOC(cudaMLO->timer,"collect_data_2");



	/*=======================================================================================
	                                   MAXIMIZATION
	=======================================================================================*/

	CTIC(cudaMLO->timer,"maximization");

	for (long int ipart = 0; ipart < sp.nr_particles; ipart++)
	{
		long int part_id = baseMLO->mydata.ori_particles[op.my_ori_particle].particles_id[ipart];
		int group_id = baseMLO->mydata.getGroupId(part_id);

		/*======================================================
		                     TRANSLATIONS
		======================================================*/

		long unsigned translation_num((sp.itrans_max - sp.itrans_min + 1) * sp.nr_oversampled_trans);

		CudaGlobalPtr<XFLOAT> trans_x(translation_num, cudaMLO->devBundle->allocator);
		CudaGlobalPtr<XFLOAT> trans_y(translation_num, cudaMLO->devBundle->allocator);
		CudaGlobalPtr<XFLOAT> trans_z(translation_num, cudaMLO->devBundle->allocator);

		int j = 0;
		for (long int itrans = 0; itrans < (sp.itrans_max - sp.itrans_min + 1); itrans++)
		{
			baseMLO->sampling.getTranslations(itrans, baseMLO->adaptive_oversampling, oversampled_translations_x,
					oversampled_translations_y, oversampled_translations_z,
					(baseMLO->do_helical_refine) && (! baseMLO->ignore_helical_symmetry), baseMLO->helical_rise_initial / baseMLO->mymodel.pixel_size, baseMLO->helical_twist_initial); //TODO Called multiple time to generate same list, reuse the same list

			for (long int iover_trans = 0; iover_trans < oversampled_translations_x.size(); iover_trans++)
			{
				RFLOAT xshift = 0., yshift = 0., zshift = 0.;

				xshift = oversampled_translations_x[iover_trans];
				yshift = oversampled_translations_y[iover_trans];
				if (cudaMLO->dataIs3D)
					zshift = oversampled_translations_z[iover_trans];

				if ( (baseMLO->do_helical_refine) && (! baseMLO->ignore_helical_symmetry) )
				{
					RFLOAT rot_deg = DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_ROT);
					RFLOAT tilt_deg = DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_TILT);
					RFLOAT psi_deg = DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_PSI);
					transformCartesianAndHelicalCoords(xshift, yshift, zshift, xshift, yshift, zshift, rot_deg, tilt_deg, psi_deg, (cudaMLO->dataIs3D) ? (3) : (2), HELICAL_TO_CART_COORDS);
				}

				trans_x[j] = -2 * PI * xshift / (double)baseMLO->mymodel.ori_size;
				trans_y[j] = -2 * PI * yshift / (double)baseMLO->mymodel.ori_size;
				trans_z[j] = -2 * PI * zshift / (double)baseMLO->mymodel.ori_size;
				j ++;
			}
		}

		trans_x.put_on_device();
		trans_y.put_on_device();
		trans_z.put_on_device();


		/*======================================================
		                     IMAGES
		======================================================*/

		CUSTOM_ALLOCATOR_REGION_NAME("TRANS_3");

		CTIC(cudaMLO->timer,"translation_3");

		CudaGlobalPtr<XFLOAT> Fimgs_real(image_size, cudaMLO->devBundle->allocator);
		CudaGlobalPtr<XFLOAT> Fimgs_imag(image_size, cudaMLO->devBundle->allocator);
		CudaGlobalPtr<XFLOAT> Fimgs_nomask_real(image_size, cudaMLO->devBundle->allocator);
		CudaGlobalPtr<XFLOAT> Fimgs_nomask_imag(image_size, cudaMLO->devBundle->allocator);

		MultidimArray<Complex > Fimg, Fimg_nonmask;
		windowFourierTransform(op.Fimgs[ipart], Fimg, sp.current_image_size);
		windowFourierTransform(op.Fimgs_nomask[ipart], Fimg_nonmask, sp.current_image_size);

		for (unsigned i = 0; i < image_size; i ++)
		{
			Fimgs_real[i] = Fimg.data[i].real;
			Fimgs_imag[i] = Fimg.data[i].imag;
			Fimgs_nomask_real[i] = Fimg_nonmask.data[i].real;
			Fimgs_nomask_imag[i] = Fimg_nonmask.data[i].imag;
		}

		Fimgs_real.put_on_device();
		Fimgs_imag.put_on_device();
		Fimgs_nomask_real.put_on_device();
		Fimgs_nomask_imag.put_on_device();

		CTOC(cudaMLO->timer,"translation_3");


		/*======================================================
		                       SCALE
		======================================================*/

		XFLOAT part_scale(1.);

		if (baseMLO->do_scale_correction)
		{
			part_scale = baseMLO->mymodel.scale_correction[group_id];
			if (part_scale > 10000.)
			{
				std::cerr << " rlnMicrographScaleCorrection= " << part_scale << " group= " << group_id + 1 << std::endl;
				CRITICAL(ERRHIGHSCALE);
			}
			else if (part_scale < 0.001)
			{
				if (!have_warned_small_scale)
				{
					std::cout << " WARNING: ignoring group " << group_id + 1 << " with very small or negative scale (" << part_scale <<
							"); Use larger groups for more stable scale estimates." << std::endl;
					have_warned_small_scale = true;
				}
				part_scale = 0.001;
			}
		}

		CudaGlobalPtr<XFLOAT> ctfs(image_size, cudaMLO->devBundle->allocator);

		if (baseMLO->do_ctf_correction)
		{
			for (unsigned i = 0; i < image_size; i++)
				ctfs[i] = (XFLOAT) op.local_Fctfs[ipart].data[i] * part_scale;
		}
		else //TODO should be handled by memset
			for (unsigned i = 0; i < image_size; i++)
				ctfs[i] = part_scale;

		ctfs.put_on_device();

		/*======================================================
		                       MINVSIGMA
		======================================================*/

		CudaGlobalPtr<XFLOAT> Minvsigma2s(image_size, cudaMLO->devBundle->allocator);

		if (baseMLO->do_map)
			for (unsigned i = 0; i < image_size; i++)
				Minvsigma2s[i] = op.local_Minvsigma2s[ipart].data[i];
		else
			for (unsigned i = 0; i < image_size; i++)
				Minvsigma2s[i] = 1;

		Minvsigma2s.put_on_device();

		/*======================================================
		                      CLASS LOOP
		======================================================*/

		CUSTOM_ALLOCATOR_REGION_NAME("wdiff2s");

		CudaGlobalPtr<XFLOAT> wdiff2s_AA(baseMLO->mymodel.nr_classes*image_size, 0, cudaMLO->devBundle->allocator);
		CudaGlobalPtr<XFLOAT> wdiff2s_XA(baseMLO->mymodel.nr_classes*image_size, 0, cudaMLO->devBundle->allocator);
		CudaGlobalPtr<XFLOAT> wdiff2s_sum(image_size, 0, cudaMLO->devBundle->allocator);

		wdiff2s_AA.device_alloc();
		wdiff2s_AA.device_init(0.f);
		wdiff2s_XA.device_alloc();
		wdiff2s_XA.device_init(0.f);

		unsigned long AAXA_pos=0;

		wdiff2s_sum.device_alloc();
		wdiff2s_sum.device_init(0.f);

		CUSTOM_ALLOCATOR_REGION_NAME("BP_data");

		// Loop from iclass_min to iclass_max to deal with seed generation in first iteration
		CudaGlobalPtr<XFLOAT> sorted_weights(ProjectionData[ipart].orientationNumAllClasses * translation_num, 0, cudaMLO->devBundle->allocator);
		std::vector<CudaGlobalPtr<XFLOAT> > eulers(baseMLO->mymodel.nr_classes, cudaMLO->devBundle->allocator);

		int classPos = 0;

		for (int exp_iclass = sp.iclass_min; exp_iclass <= sp.iclass_max; exp_iclass++)
			DEBUG_HANDLE_ERROR(hipStreamSynchronize(cudaMLO->classStreams[exp_iclass]));
		DEBUG_HANDLE_ERROR(hipStreamSynchronize(hipStreamPerThread));

		for (int exp_iclass = sp.iclass_min; exp_iclass <= sp.iclass_max; exp_iclass++)
		{
			if((baseMLO->mymodel.pdf_class[exp_iclass] == 0.) || (ProjectionData[ipart].class_entries[exp_iclass] == 0))
				continue;

			// Use the constructed mask to construct a partial class-specific input
			IndexedDataArray thisClassFinePassWeights(FinePassWeights[ipart],FPCMasks[ipart][exp_iclass], cudaMLO->devBundle->allocator);

			CTIC(cudaMLO->timer,"thisClassProjectionSetupCoarse");
			// use "slice" constructor with class-specific parameters to retrieve a temporary ProjectionParams with data for this class
			ProjectionParams thisClassProjectionData(	ProjectionData[ipart],
														ProjectionData[ipart].class_idx[exp_iclass],
														ProjectionData[ipart].class_idx[exp_iclass]+ProjectionData[ipart].class_entries[exp_iclass]);

			thisClassProjectionData.orientation_num[0] = ProjectionData[ipart].orientation_num[exp_iclass];
			CTOC(cudaMLO->timer,"thisClassProjectionSetupCoarse");

			long unsigned orientation_num(thisClassProjectionData.orientation_num[0]);

			/*======================================================
								PROJECTIONS
			======================================================*/

			eulers[exp_iclass].setSize(orientation_num * 9);
			eulers[exp_iclass].setStream(cudaMLO->classStreams[exp_iclass]);
			eulers[exp_iclass].host_alloc();

			CTIC(cudaMLO->timer,"generateEulerMatricesProjector");

			generateEulerMatrices(
					baseMLO->mymodel.PPref[exp_iclass].padding_factor,
					thisClassProjectionData,
					&eulers[exp_iclass][0],
					!IS_NOT_INV);

			eulers[exp_iclass].device_alloc();
			eulers[exp_iclass].cp_to_device();

			CTOC(cudaMLO->timer,"generateEulerMatricesProjector");


			/*======================================================
								 MAP WEIGHTS
			======================================================*/

			CTIC(cudaMLO->timer,"pre_wavg_map");

			for (long unsigned i = 0; i < orientation_num*translation_num; i++)
				sorted_weights[classPos+i] = -999.;

			for (long unsigned i = 0; i < thisClassFinePassWeights.weights.getSize(); i++)
				sorted_weights[classPos+(thisClassFinePassWeights.rot_idx[i]) * translation_num + thisClassFinePassWeights.trans_idx[i] ]
								= thisClassFinePassWeights.weights[i];

			classPos+=orientation_num*translation_num;
			CTOC(cudaMLO->timer,"pre_wavg_map");
		}
		sorted_weights.put_on_device();

		// These syncs are necessary (for multiple ranks on the same GPU), and (assumed) low-cost.
		for (int exp_iclass = sp.iclass_min; exp_iclass <= sp.iclass_max; exp_iclass++)
			DEBUG_HANDLE_ERROR(hipStreamSynchronize(cudaMLO->classStreams[exp_iclass]));
		DEBUG_HANDLE_ERROR(hipStreamSynchronize(hipStreamPerThread));

		classPos = 0;
		for (int exp_iclass = sp.iclass_min; exp_iclass <= sp.iclass_max; exp_iclass++)
		{
			if((baseMLO->mymodel.pdf_class[exp_iclass] == 0.) || (ProjectionData[ipart].class_entries[exp_iclass] == 0))
			continue;
			/*======================================================
								 KERNEL CALL
			======================================================*/

			long unsigned orientation_num(ProjectionData[ipart].orientation_num[exp_iclass]);

			CudaProjectorKernel projKernel = CudaProjectorKernel::makeKernel(
					cudaMLO->devBundle->cudaProjectors[exp_iclass],
					op.local_Minvsigma2s[0].xdim,
					op.local_Minvsigma2s[0].ydim,
					op.local_Minvsigma2s[0].zdim,
					op.local_Minvsigma2s[0].xdim-1);

			runWavgKernel(
					projKernel,
					~eulers[exp_iclass],
					~Fimgs_real,
					~Fimgs_imag,
					~trans_x,
					~trans_y,
					~trans_z,
					&sorted_weights.d_ptr[classPos],
					~ctfs,
					~wdiff2s_sum,
					&wdiff2s_AA(AAXA_pos),
					&wdiff2s_XA(AAXA_pos),
					op,
					orientation_num,
					translation_num,
					image_size,
					ipart,
					group_id,
					exp_iclass,
					part_scale,
					baseMLO->refs_are_ctf_corrected,
					cudaMLO->dataIs3D,
					cudaMLO->classStreams[exp_iclass]);

			/*======================================================
								BACKPROJECTION
			======================================================*/

#ifdef TIMING
			if (op.my_ori_particle == baseMLO->exp_my_first_ori_particle)
				baseMLO->timer.tic(baseMLO->TIMING_WSUM_BACKPROJ);
#endif

			CTIC(cudaMLO->timer,"backproject");

			runBackProjectKernel(
				cudaMLO->devBundle->cudaBackprojectors[exp_iclass],
				projKernel,
				~Fimgs_nomask_real,
				~Fimgs_nomask_imag,
				~trans_x,
				~trans_y,
				~trans_z,
				&sorted_weights.d_ptr[classPos],
				~Minvsigma2s,
				~ctfs,
				translation_num,
				(XFLOAT) op.significant_weight[ipart],
				(XFLOAT) op.sum_weight[ipart],
				~eulers[exp_iclass],
				op.local_Minvsigma2s[0].xdim,
				op.local_Minvsigma2s[0].ydim,
				op.local_Minvsigma2s[0].zdim,
				orientation_num,
				cudaMLO->dataIs3D,
				baseMLO->do_sgd,
				cudaMLO->classStreams[exp_iclass]);

			CTOC(cudaMLO->timer,"backproject");

#ifdef TIMING
			if (op.my_ori_particle == baseMLO->exp_my_first_ori_particle)
				baseMLO->timer.toc(baseMLO->TIMING_WSUM_BACKPROJ);
#endif

			//Update indices
			AAXA_pos += image_size;
			classPos += orientation_num*translation_num;

		} // end loop iclass

		CUSTOM_ALLOCATOR_REGION_NAME("UNSET");

		// NOTE: We've never seen that this sync is necessary, but it is needed in principle, and
		// its absence in other parts of the code has caused issues. It is also very low-cost.
		for (int exp_iclass = sp.iclass_min; exp_iclass <= sp.iclass_max; exp_iclass++)
			DEBUG_HANDLE_ERROR(hipStreamSynchronize(cudaMLO->classStreams[exp_iclass]));
		DEBUG_HANDLE_ERROR(hipStreamSynchronize(hipStreamPerThread));

		wdiff2s_AA.cp_to_host();
		wdiff2s_XA.cp_to_host();
		wdiff2s_sum.cp_to_host();
		DEBUG_HANDLE_ERROR(hipStreamSynchronize(hipStreamPerThread));

		AAXA_pos=0;

		for (int exp_iclass = sp.iclass_min; exp_iclass <= sp.iclass_max; exp_iclass++)
		{
			if((baseMLO->mymodel.pdf_class[exp_iclass] == 0.) || (ProjectionData[ipart].class_entries[exp_iclass] == 0))
				continue;
			for (long int j = 0; j < image_size; j++)
			{
				int ires = DIRECT_MULTIDIM_ELEM(baseMLO->Mresol_fine, j);
				if (ires > -1 && baseMLO->do_scale_correction &&
						DIRECT_A1D_ELEM(baseMLO->mymodel.data_vs_prior_class[exp_iclass], ires) > 3.)
				{
					DIRECT_A1D_ELEM(exp_wsum_scale_correction_AA[ipart], ires) += wdiff2s_AA[AAXA_pos+j];
					DIRECT_A1D_ELEM(exp_wsum_scale_correction_XA[ipart], ires) += wdiff2s_XA[AAXA_pos+j];
				}
			}
			AAXA_pos += image_size;
		} // end loop iclass
		for (long int j = 0; j < image_size; j++)
		{
			int ires = DIRECT_MULTIDIM_ELEM(baseMLO->Mresol_fine, j);
			if (ires > -1)
			{
				thr_wsum_sigma2_noise[group_id].data[ires] += (RFLOAT) wdiff2s_sum[j];
				exp_wsum_norm_correction[ipart] += (RFLOAT) wdiff2s_sum[j]; //TODO could be gpu-reduced
			}
		}
	} // end loop ipart
	CTOC(cudaMLO->timer,"maximization");


	CTIC(cudaMLO->timer,"store_post_gpu");

	// Extend norm_correction and sigma2_noise estimation to higher resolutions for all particles
	// Also calculate dLL for each particle and store in metadata
	// loop over all particles inside this ori_particle
	RFLOAT thr_avg_norm_correction = 0.;
	RFLOAT thr_sum_dLL = 0., thr_sum_Pmax = 0.;
	for (long int ipart = 0; ipart < sp.nr_particles; ipart++)
	{
		long int part_id = baseMLO->mydata.ori_particles[op.my_ori_particle].particles_id[ipart];
		int group_id = baseMLO->mydata.getGroupId(part_id);

		// If the current images were smaller than the original size, fill the rest of wsum_model.sigma2_noise with the power_class spectrum of the images
		for (int ires = baseMLO->mymodel.current_size/2 + 1; ires < baseMLO->mymodel.ori_size/2 + 1; ires++)
		{
			DIRECT_A1D_ELEM(thr_wsum_sigma2_noise[group_id], ires) += DIRECT_A1D_ELEM(op.power_imgs[ipart], ires);
			// Also extend the weighted sum of the norm_correction
			exp_wsum_norm_correction[ipart] += DIRECT_A1D_ELEM(op.power_imgs[ipart], ires);
		}

		// Store norm_correction
		// Multiply by old value because the old norm_correction term was already applied to the image
		if (baseMLO->do_norm_correction)
		{
			RFLOAT old_norm_correction = DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_NORM);
			old_norm_correction /= baseMLO->mymodel.avg_norm_correction;
			// The factor two below is because exp_wsum_norm_correctiom is similar to sigma2_noise, which is the variance for the real/imag components
			// The variance of the total image (on which one normalizes) is twice this value!
			RFLOAT normcorr = old_norm_correction * sqrt(exp_wsum_norm_correction[ipart] * 2.);
			thr_avg_norm_correction += normcorr;

			// Now set the new norm_correction in the relevant position of exp_metadata
			DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_NORM) = normcorr;


			// Print warning for strange norm-correction values
			if (!((baseMLO->iter == 1 && baseMLO->do_firstiter_cc) || baseMLO->do_always_cc) && DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_NORM) > 10.)
			{
				std::cout << " WARNING: norm_correction= "<< DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_NORM)
						<< " for particle " << part_id << " in group " << group_id + 1
						<< "; Are your groups large enough? Or is the reference on the correct greyscale?" << std::endl;
			}

		}

		// Store weighted sums for scale_correction
		if (baseMLO->do_scale_correction)
		{
			// Divide XA by the old scale_correction and AA by the square of that, because was incorporated into Fctf
			exp_wsum_scale_correction_XA[ipart] /= baseMLO->mymodel.scale_correction[group_id];
			exp_wsum_scale_correction_AA[ipart] /= baseMLO->mymodel.scale_correction[group_id] * baseMLO->mymodel.scale_correction[group_id];

			thr_wsum_signal_product_spectra[group_id] += exp_wsum_scale_correction_XA[ipart];
			thr_wsum_reference_power_spectra[group_id] += exp_wsum_scale_correction_AA[ipart];
		}

		// Calculate DLL for each particle
		RFLOAT logsigma2 = 0.;
		FOR_ALL_DIRECT_ELEMENTS_IN_MULTIDIMARRAY(baseMLO->Mresol_fine)
		{
			int ires = DIRECT_MULTIDIM_ELEM(baseMLO->Mresol_fine, n);
			// Note there is no sqrt in the normalisation term because of the 2-dimensionality of the complex-plane
			// Also exclude origin from logsigma2, as this will not be considered in the P-calculations
			if (ires > 0)
				logsigma2 += log( 2. * PI * DIRECT_A1D_ELEM(baseMLO->mymodel.sigma2_noise[group_id], ires));
		}
		RFLOAT dLL;

		XFLOAT local_norm = (XFLOAT)op.avg_diff2[ipart];
		if (local_norm - op.min_diff2[ipart] > 50)
			local_norm = op.min_diff2[ipart] + 50;

		if ((baseMLO->iter==1 && baseMLO->do_firstiter_cc) || baseMLO->do_always_cc)
			dLL = -op.min_diff2[ipart];
		else
			dLL = log(op.sum_weight[ipart]) - local_norm - logsigma2;

		// Store dLL of each image in the output array, and keep track of total sum
		DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_DLL) = dLL;
		thr_sum_dLL += dLL;

		// Also store sum of Pmax
		thr_sum_Pmax += DIRECT_A2D_ELEM(baseMLO->exp_metadata, op.metadata_offset + ipart, METADATA_PMAX);

	}

	// Now, inside a global_mutex, update the other weighted sums among all threads
	if (!baseMLO->do_skip_maximization)
	{
		pthread_mutex_lock(&global_mutex);
		for (int n = 0; n < baseMLO->mymodel.nr_groups; n++)
		{
			baseMLO->wsum_model.sigma2_noise[n] += thr_wsum_sigma2_noise[n];
			baseMLO->wsum_model.sumw_group[n] += thr_sumw_group[n];
			if (baseMLO->do_scale_correction)
			{
				baseMLO->wsum_model.wsum_signal_product_spectra[n] += thr_wsum_signal_product_spectra[n];
				baseMLO->wsum_model.wsum_reference_power_spectra[n] += thr_wsum_reference_power_spectra[n];
			}
		}
		for (int n = 0; n < baseMLO->mymodel.nr_classes; n++)
		{
			baseMLO->wsum_model.pdf_class[n] += thr_wsum_pdf_class[n];
			if (baseMLO->mymodel.ref_dim == 2)
			{
				XX(baseMLO->wsum_model.prior_offset_class[n]) += thr_wsum_prior_offsetx_class[n];
				YY(baseMLO->wsum_model.prior_offset_class[n]) += thr_wsum_prior_offsety_class[n];
			}

			if (!(baseMLO->do_skip_align || baseMLO->do_skip_rotate) )
				baseMLO->wsum_model.pdf_direction[n] += thr_wsum_pdf_direction[n];
		}
		baseMLO->wsum_model.sigma2_offset += thr_wsum_sigma2_offset;
		if (baseMLO->do_norm_correction)
			baseMLO->wsum_model.avg_norm_correction += thr_avg_norm_correction;
		baseMLO->wsum_model.LL += thr_sum_dLL;
		baseMLO->wsum_model.ave_Pmax += thr_sum_Pmax;
		pthread_mutex_unlock(&global_mutex);
	} // end if !do_skip_maximization

	CTOC(cudaMLO->timer,"store_post_gpu");
#ifdef TIMING
	if (op.my_ori_particle == baseMLO->exp_my_first_ori_particle)
		baseMLO->timer.toc(baseMLO->TIMING_ESP_WSUM);
#endif
}
size_t MlDeviceBundle::checkFixedSizedObjects(int shares)
{
	int devCount;
	size_t BoxLimit;
	HANDLE_ERROR(hipGetDeviceCount(&devCount));
	if(device_id >= devCount)
		CRITICAL(ERR_GPUID);

	HANDLE_ERROR(hipSetDevice(device_id));

	size_t free(0), total(0);
	DEBUG_HANDLE_ERROR(hipMemGetInfo( &free, &total ));
	float margin(1.05);
	BoxLimit = pow(free/(margin*2.5*sizeof(XFLOAT)*((float)shares)),(1/3.0)) / (2.0);
	size_t BytesNeeded = ((float)shares)*margin*2.5*sizeof(XFLOAT)*pow((baseMLO->mymodel.ori_size*2),3);

	return(BoxLimit);
}
void MlDeviceBundle::setupFixedSizedObjects()
{
	unsigned nr_classes = baseMLO->mymodel.nr_classes;

	int devCount;
	HANDLE_ERROR(hipGetDeviceCount(&devCount));
	if(device_id >= devCount)
	{
		//std::cerr << " using device_id=" << device_id << " (device no. " << device_id+1 << ") which is higher than the available number of devices=" << devCount << std::endl;
		CRITICAL(ERR_GPUID);
	}
	else
		HANDLE_ERROR(hipSetDevice(device_id));

	//Can we pre-generate projector plan and corresponding euler matrices for all particles
	if (baseMLO->do_skip_align || baseMLO->do_skip_rotate || baseMLO->do_auto_refine || baseMLO->mymodel.orientational_prior_mode != NOPRIOR)
		generateProjectionPlanOnTheFly = true;
	else
		generateProjectionPlanOnTheFly = false;

	// clear() called on std::vector appears to set size=0, even if we have an explicit
	// destructor for each member, so we need to set the size to what is was before
	cudaProjectors.resize(nr_classes);
	cudaBackprojectors.resize(nr_classes);

	/*======================================================
	              PROJECTOR AND BACKPROJECTOR
	======================================================*/

	//Loop over classes
	for (int iclass = 0; iclass < nr_classes; iclass++)
	{
		cudaProjectors[iclass].setMdlDim(
				baseMLO->mymodel.PPref[iclass].data.xdim,
				baseMLO->mymodel.PPref[iclass].data.ydim,
				baseMLO->mymodel.PPref[iclass].data.zdim,
				baseMLO->mymodel.PPref[iclass].data.yinit,
				baseMLO->mymodel.PPref[iclass].data.zinit,
				baseMLO->mymodel.PPref[iclass].r_max,
				baseMLO->mymodel.PPref[iclass].padding_factor);

		cudaProjectors[iclass].initMdl(baseMLO->mymodel.PPref[iclass].data.data);

		cudaBackprojectors[iclass].setMdlDim(
				baseMLO->wsum_model.BPref[iclass].data.xdim,
				baseMLO->wsum_model.BPref[iclass].data.ydim,
				baseMLO->wsum_model.BPref[iclass].data.zdim,
				baseMLO->wsum_model.BPref[iclass].data.yinit,
				baseMLO->wsum_model.BPref[iclass].data.zinit,
				baseMLO->wsum_model.BPref[iclass].r_max,
				baseMLO->wsum_model.BPref[iclass].padding_factor);

		cudaBackprojectors[iclass].initMdl();
	}

	/*======================================================
	                    CUSTOM ALLOCATOR
	======================================================*/

	int memAlignmentSize;
	hipDeviceGetAttribute ( &memAlignmentSize, hipDeviceAttributeTextureAlignment, device_id );
	allocator = new CudaCustomAllocator(0, memAlignmentSize);
}

void MlDeviceBundle::setupTunableSizedObjects(size_t allocationSize)
{
	unsigned nr_classes = baseMLO->mymodel.nr_classes;
	int devCount;
	HANDLE_ERROR(hipGetDeviceCount(&devCount));
	if(device_id >= devCount)
	{
		//std::cerr << " using device_id=" << device_id << " (device no. " << device_id+1 << ") which is higher than the available number of devices=" << devCount << std::endl;
		CRITICAL(ERR_GPUID);
	}
	else
		HANDLE_ERROR(hipSetDevice(device_id));

	/*======================================================
	                    CUSTOM ALLOCATOR
	======================================================*/
#ifdef DEBUG_CUDA
	printf("DEBUG: Total GPU allocation size set to %zu MB on device id %d.\n", allocationSize / (1000*1000), device_id);
#endif
#ifndef CUDA_NO_CUSTOM_ALLOCATION
	allocator->resize(allocationSize);
#endif


	/*======================================================
	                    PROJECTION PLAN
	======================================================*/

	coarseProjectionPlans.resize(nr_classes, allocator);

	for (int iclass = 0; iclass < nr_classes; iclass++)
	{
		//If doing predefined projector plan at all and is this class significant
		if (!generateProjectionPlanOnTheFly && baseMLO->mymodel.pdf_class[iclass] > 0.)
		{
			std::vector<int> exp_pointer_dir_nonzeroprior;
			std::vector<int> exp_pointer_psi_nonzeroprior;
			std::vector<RFLOAT> exp_directions_prior;
			std::vector<RFLOAT> exp_psi_prior;

			long unsigned itrans_max = baseMLO->sampling.NrTranslationalSamplings() - 1;
			long unsigned nr_idir = baseMLO->sampling.NrDirections(0, &exp_pointer_dir_nonzeroprior);
			long unsigned nr_ipsi = baseMLO->sampling.NrPsiSamplings(0, &exp_pointer_psi_nonzeroprior );

			coarseProjectionPlans[iclass].setup(
					baseMLO->sampling,
					exp_directions_prior,
					exp_psi_prior,
					exp_pointer_dir_nonzeroprior,
					exp_pointer_psi_nonzeroprior,
					NULL, //Mcoarse_significant
					baseMLO->mymodel.pdf_class,
					baseMLO->mymodel.pdf_direction,
					nr_idir,
					nr_ipsi,
					0, //idir_min
					nr_idir - 1, //idir_max
					0, //ipsi_min
					nr_ipsi - 1, //ipsi_max
					0, //itrans_min
					itrans_max,
					0, //current_oversampling
					1, //nr_oversampled_rot
					iclass,
					true, //coarse
					!IS_NOT_INV,
					baseMLO->do_skip_align,
					baseMLO->do_skip_rotate,
					baseMLO->mymodel.orientational_prior_mode
					);
		}
	}
};

void MlOptimiserCuda::resetData()
{
	int devCount;
	HANDLE_ERROR(hipGetDeviceCount(&devCount));
	if(device_id >= devCount)
	{
		//std::cerr << " using device_id=" << device_id << " (device no. " << device_id+1 << ") which is higher than the available number of devices=" << devCount << std::endl;
		CRITICAL(ERR_GPUID);
	}
	else
		HANDLE_ERROR(hipSetDevice(device_id));

	unsigned nr_classes = baseMLO->mymodel.nr_classes;

	classStreams.resize(nr_classes, 0);
	for (int i = 0; i < nr_classes; i++)
		HANDLE_ERROR(hipStreamCreate(&classStreams[i])); //HANDLE_ERROR(hipStreamCreateWithFlags(&classStreams[i],hipStreamNonBlocking));

	transformer1.clear();
	transformer2.clear();

	failsafe_attempts = 0;
};

void MlOptimiserCuda::doThreadExpectationSomeParticles(int thread_id)
{
#ifdef TIMING
	// Only time one thread
	if (thread_id == 0)
		baseMLO->timer.tic(baseMLO->TIMING_ESP_THR);
#endif
//	CTOC(cudaMLO->timer,"interParticle");

	int devCount;
	HANDLE_ERROR(hipGetDeviceCount(&devCount));
	if(device_id >= devCount)
	{
		//std::cerr << " using device_id=" << device_id << " (device no. " << device_id+1 << ") which is higher than the available number of devices=" << devCount << std::endl;
		CRITICAL(ERR_GPUID);
	}
	else
		DEBUG_HANDLE_ERROR(hipSetDevice(device_id));
	//std::cerr << " calling on device " << device_id << std::endl;
	//put mweight allocation here
	size_t first_ipart = 0, last_ipart = 0;

	while (baseMLO->exp_ipart_ThreadTaskDistributor->getTasks(first_ipart, last_ipart))
	{
		CTIC(timer,"oneTask");
		for (long unsigned ipart = first_ipart; ipart <= last_ipart; ipart++)
		{
			CTIC(timer,"oneParticle");
#ifdef TIMING
	// Only time one thread
	if (thread_id == 0)
		baseMLO->timer.tic(baseMLO->TIMING_ESP_DIFF2_A);
#endif
			unsigned my_ori_particle = baseMLO->exp_my_first_ori_particle + ipart;
			SamplingParameters sp;
			sp.nr_particles = baseMLO->mydata.ori_particles[my_ori_particle].particles_id.size();

			OptimisationParamters op(sp.nr_particles, my_ori_particle);

			// In the first iteration, multiple seeds will be generated
			// A single random class is selected for each pool of images, and one does not marginalise over the orientations
			// The optimal orientation is based on signal-product (rather than the signal-intensity sensitive Gaussian)
			// If do_firstiter_cc, then first perform a single iteration with K=1 and cross-correlation criteria, afterwards

			// Decide which classes to integrate over (for random class assignment in 1st iteration)
			sp.iclass_min = 0;
			sp.iclass_max = baseMLO->mymodel.nr_classes - 1;
			// low-pass filter again and generate the seeds
			if (baseMLO->do_generate_seeds)
			{
				if (baseMLO->do_firstiter_cc && baseMLO->iter == 1)
				{
					// In first (CC) iter, use a single reference (and CC)
					sp.iclass_min = sp.iclass_max = 0;
				}
				else if ( (baseMLO->do_firstiter_cc && baseMLO->iter == 2) ||

						(!baseMLO->do_firstiter_cc && baseMLO->iter == 1))
				{
					// In second CC iter, or first iter without CC: generate the seeds
					// Now select a single random class
					// exp_part_id is already in randomized order (controlled by -seed)
					// WARNING: USING SAME iclass_min AND iclass_max FOR SomeParticles!!
		    		// Make sure random division is always the same with the same seed
					long int idx = my_ori_particle - baseMLO->exp_my_first_ori_particle;
					if (idx >= baseMLO->exp_random_class_some_particles.size())
						REPORT_ERROR("BUG: expectationOneParticle idx>random_class_some_particles.size()");
					sp.iclass_min = sp.iclass_max = baseMLO->exp_random_class_some_particles[idx];
				}
			}

			// Global exp_metadata array has metadata of all ori_particles. Where does my_ori_particle start?
			for (long int iori = baseMLO->exp_my_first_ori_particle; iori <= baseMLO->exp_my_last_ori_particle; iori++)
			{
				if (iori == my_ori_particle) break;
				op.metadata_offset += baseMLO->mydata.ori_particles[iori].particles_id.size();
			}
#ifdef TIMING
	// Only time one thread
	if (thread_id == 0)
		baseMLO->timer.toc(baseMLO->TIMING_ESP_DIFF2_A);
#endif
			CTIC(timer,"getFourierTransformsAndCtfs");
			getFourierTransformsAndCtfs(my_ori_particle, op, sp, baseMLO, this);
			CTOC(timer,"getFourierTransformsAndCtfs");

			if (baseMLO->do_realign_movies && baseMLO->movie_frame_running_avg_side > 0)
			{
				baseMLO->calculateRunningAveragesOfMovieFrames(my_ori_particle, op.Fimgs, op.power_imgs, op.highres_Xi2_imgs);
			}

			// To deal with skipped alignments/rotations
			if (baseMLO->do_skip_align)
			{
				sp.itrans_min = sp.itrans_max = sp.idir_min = sp.idir_max = sp.ipsi_min = sp.ipsi_max =
						my_ori_particle - baseMLO->exp_my_first_ori_particle;
			}
			else
			{
				sp.itrans_min = 0;
				sp.itrans_max = baseMLO->sampling.NrTranslationalSamplings() - 1;

				if (baseMLO->do_skip_rotate)
				{
					sp.idir_min = sp.idir_max = sp.ipsi_min = sp.ipsi_max =
							my_ori_particle - baseMLO->exp_my_first_ori_particle;
				}
				else
				{
					sp.idir_min = sp.ipsi_min = 0;
					sp.idir_max = baseMLO->sampling.NrDirections(0, &op.pointer_dir_nonzeroprior) - 1;
					sp.ipsi_max = baseMLO->sampling.NrPsiSamplings(0, &op.pointer_psi_nonzeroprior ) - 1;
				}
			}

			// Initialise significant weight to minus one, so that all coarse sampling points will be handled in the first pass
			op.significant_weight.resize(sp.nr_particles, -1.);

			// Only perform a second pass when using adaptive oversampling
			//int nr_sampling_passes = (baseMLO->adaptive_oversampling > 0) ? 2 : 1;
			// But on the gpu the data-structures are different between passes, so we need to make a symbolic pass to set the weights up for storeWS
			int nr_sampling_passes = 2;

			/// -- This is a iframe-indexed vector, each entry of which is a dense data-array. These are replacements to using
			//    Mweight in the sparse (Fine-sampled) pass, coarse is unused but created empty input for convert ( FIXME )
			std::vector <IndexedDataArray> CoarsePassWeights(1, devBundle->allocator) ,FinePassWeights(sp.nr_particles, devBundle->allocator);
			// -- This is a iframe-indexed vector, each entry of which is a class-indexed vector of masks, one for each
			//    class in FinePassWeights
			std::vector < std::vector <IndexedDataArrayMask> > FinePassClassMasks(sp.nr_particles, std::vector <IndexedDataArrayMask>(baseMLO->mymodel.nr_classes, devBundle->allocator));
			// -- This is a iframe-indexed vector, each entry of which is parameters used in the projection-operations *after* the
			//    coarse pass, declared here to keep scope to storeWS
			std::vector < ProjectionParams > FineProjectionData(sp.nr_particles, baseMLO->mymodel.nr_classes);

			std::vector < cudaStager<unsigned long> > stagerD2(sp.nr_particles,devBundle->allocator), stagerSWS(sp.nr_particles,devBundle->allocator);

			for (int ipass = 0; ipass < nr_sampling_passes; ipass++)
			{
				CTIC(timer,"weightPass");
#ifdef TIMING
	// Only time one thread
	if (thread_id == 0)
		baseMLO->timer.tic(baseMLO->TIMING_ESP_DIFF2_B);
#endif
				if (baseMLO->strict_highres_exp > 0.)
					// Use smaller images in both passes and keep a maximum on coarse_size, just like in FREALIGN
					sp.current_image_size = baseMLO->coarse_size;
				else if (baseMLO->adaptive_oversampling > 0)
					// Use smaller images in the first pass, larger ones in the second pass
					sp.current_image_size = (ipass == 0) ? baseMLO->coarse_size : baseMLO->mymodel.current_size;
				else
					sp.current_image_size = baseMLO->mymodel.current_size;

				// Use coarse sampling in the first pass, oversampled one the second pass
				sp.current_oversampling = (ipass == 0) ? 0 : baseMLO->adaptive_oversampling;

				sp.nr_dir = (baseMLO->do_skip_align || baseMLO->do_skip_rotate) ? 1 : baseMLO->sampling.NrDirections(0, &op.pointer_dir_nonzeroprior);
				sp.nr_psi = (baseMLO->do_skip_align || baseMLO->do_skip_rotate) ? 1 : baseMLO->sampling.NrPsiSamplings(0, &op.pointer_psi_nonzeroprior);
				sp.nr_trans = (baseMLO->do_skip_align) ? 1 : baseMLO->sampling.NrTranslationalSamplings();
				sp.nr_oversampled_rot = baseMLO->sampling.oversamplingFactorOrientations(sp.current_oversampling);
				sp.nr_oversampled_trans = baseMLO->sampling.oversamplingFactorTranslations(sp.current_oversampling);
#ifdef TIMING
	// Only time one thread
	if (thread_id == 0)
		baseMLO->timer.toc(baseMLO->TIMING_ESP_DIFF2_B);
#endif

				op.min_diff2.resize(sp.nr_particles, 0);
				op.avg_diff2.resize(sp.nr_particles, 0);

				if (ipass == 0)
				{
					unsigned long weightsPerPart(baseMLO->mymodel.nr_classes * sp.nr_dir * sp.nr_psi * sp.nr_trans * sp.nr_oversampled_rot * sp.nr_oversampled_trans);

					op.Mweight.resizeNoCp(1,1,sp.nr_particles, weightsPerPart);

					CudaGlobalPtr<XFLOAT> Mweight(devBundle->allocator);
					Mweight.setSize(sp.nr_particles * weightsPerPart);
					Mweight.setHstPtr(op.Mweight.data);
					Mweight.device_alloc();
					deviceInitValue<XFLOAT>(Mweight, -999.);
					Mweight.streamSync();

					CTIC(timer,"getAllSquaredDifferencesCoarse");
					getAllSquaredDifferencesCoarse(ipass, op, sp, baseMLO, this, Mweight);
					CTOC(timer,"getAllSquaredDifferencesCoarse");

					try
					{
						CTIC(timer,"convertAllSquaredDifferencesToWeightsCoarse");
						convertAllSquaredDifferencesToWeights<XFLOAT>(ipass, op, sp, baseMLO, this, CoarsePassWeights, FinePassClassMasks, Mweight);
						CTOC(timer,"convertAllSquaredDifferencesToWeightsCoarse");
					}
					catch (RelionError XE)
					{
						getAllSquaredDifferencesCoarse(ipass, op, sp, baseMLO, this, Mweight);
#ifndef CUDA_DOUBLE_PRECISION
						try {
							convertAllSquaredDifferencesToWeights<double>(ipass, op, sp, baseMLO, this, CoarsePassWeights, FinePassClassMasks, Mweight);
						}
						catch (RelionError XE)
#endif
						{
							if (failsafe_attempts > baseMLO->failsafe_threshold)
								CRITICAL(ERRNUMFAILSAFE);

							//Rerun in fail-safe mode
							convertAllSquaredDifferencesToWeights<XFLOAT>(ipass, op, sp, baseMLO, this, CoarsePassWeights, FinePassClassMasks, Mweight, true);

							if (failsafe_attempts <= 10)
							{
								std::cerr << std::endl << "WARNING: Exception (" << XE.msg << ") handled by switching to fail-safe mode." << std::endl;

								if (failsafe_attempts == 10)
									std::cerr << "NOTE: No more fail-safe warnings will be issued." << std::endl;
							}

							failsafe_attempts ++;
						}
					}
				}
				else
				{
#ifdef TIMING
	// Only time one thread
	if (thread_id == 0)
		baseMLO->timer.tic(baseMLO->TIMING_ESP_DIFF2_D);
#endif
//					// -- go through all classes and generate projectionsetups for all classes - to be used in getASDF and storeWS below --
//					// the reason to do this globally is subtle - we want the orientation_num of all classes to estimate a largest possible
//					// weight-array, which would be insanely much larger than necessary if we had to assume the worst.
					for (long int iframe = 0; iframe < sp.nr_particles; iframe++)
					{
						FineProjectionData[iframe].orientationNumAllClasses = 0;
						for (int exp_iclass = sp.iclass_min; exp_iclass <= sp.iclass_max; exp_iclass++)
						{
							if(exp_iclass>0)
								FineProjectionData[iframe].class_idx[exp_iclass] = FineProjectionData[iframe].rots.size();
							FineProjectionData[iframe].class_entries[exp_iclass] = 0;

							CTIC(timer,"generateProjectionSetup");
							FineProjectionData[iframe].orientationNumAllClasses += generateProjectionSetupFine(
									op,
									sp,
									baseMLO,
									exp_iclass,
									FineProjectionData[iframe]);
							CTOC(timer,"generateProjectionSetup");

						}
						//set a maximum possible size for all weights (to be reduced by significance-checks)
						FinePassWeights[iframe].setDataSize(FineProjectionData[iframe].orientationNumAllClasses*sp.nr_trans*sp.nr_oversampled_trans);
						FinePassWeights[iframe].dual_alloc_all();
						stagerD2[iframe].size= 2*(FineProjectionData[iframe].orientationNumAllClasses*sp.nr_trans*sp.nr_oversampled_trans);
						stagerD2[iframe].prepare();
					}
#ifdef TIMING
	// Only time one thread
	if (thread_id == 0)
		baseMLO->timer.toc(baseMLO->TIMING_ESP_DIFF2_D);
#endif
//					printf("Allocator used space before 'getAllSquaredDifferencesFine': %.2f MiB\n", (float)devBundle->allocator->getTotalUsedSpace()/(1024.*1024.));

					CTIC(timer,"getAllSquaredDifferencesFine");
					getAllSquaredDifferencesFine(ipass, op, sp, baseMLO, this, FinePassWeights, FinePassClassMasks, FineProjectionData, stagerD2);
					CTOC(timer,"getAllSquaredDifferencesFine");
					FinePassWeights[0].weights.cp_to_host();
					CudaGlobalPtr<XFLOAT> Mweight(devBundle->allocator); //DUMMY

					CTIC(timer,"convertAllSquaredDifferencesToWeightsFine");
					convertAllSquaredDifferencesToWeights<XFLOAT>(ipass, op, sp, baseMLO, this, FinePassWeights, FinePassClassMasks, Mweight);
					CTOC(timer,"convertAllSquaredDifferencesToWeightsFine");

				}

				CTOC(timer,"weightPass");
			}
#ifdef TIMING
	// Only time one thread
	if (thread_id == 0)
		baseMLO->timer.tic(baseMLO->TIMING_ESP_DIFF2_E);
#endif

			// For the reconstruction step use mymodel.current_size!
			sp.current_image_size = baseMLO->mymodel.current_size;
			for (long int iframe = 0; iframe < sp.nr_particles; iframe++)
			{
				stagerSWS[iframe].size= 2*(FineProjectionData[iframe].orientationNumAllClasses);
				stagerSWS[iframe].prepare();
			}
#ifdef TIMING
	// Only time one thread
	if (thread_id == 0)
		baseMLO->timer.toc(baseMLO->TIMING_ESP_DIFF2_E);
#endif
			CTIC(timer,"storeWeightedSums");
			storeWeightedSums(op, sp, baseMLO, this, FinePassWeights, FineProjectionData, FinePassClassMasks, stagerSWS);
			CTOC(timer,"storeWeightedSums");

			CTOC(timer,"oneParticle");
		}
		CTOC(timer,"oneTask");
	}

//	CTIC(cudaMLO->timer,"interParticle");
//	exit(0);

#ifdef TIMING
	// Only time one thread
	if (thread_id == 0)
		baseMLO->timer.toc(baseMLO->TIMING_ESP_THR);
#endif
}

