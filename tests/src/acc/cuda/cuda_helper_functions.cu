#undef ALTCPU
#include <hip/hip_runtime.h>
#include "src/ml_optimiser.h"
#include "src/acc/acc_ptr.h"
#include "src/acc/acc_projector.h"
#include "src/acc/acc_projector_plan.h"
#include "src/acc/acc_backprojector.h"
#include "src/acc/cuda/cuda_settings.h"
#include "src/acc/cuda/cuda_fft.h"
#include "src/acc/cuda/cuda_kernels/cuda_device_utils.cuh"

#ifdef CUDA_FORCESTL
#include "src/acc/cuda/cuda_utils_stl.cuh"
#else
#include "src/acc/cuda/cuda_utils_cub.cuh"
#endif

#include "src/acc/utilities.h"
#include "src/acc/acc_helper_functions.h"
#include "src/acc/cuda/cuda_kernels/BP.cuh"
#include "src/macros.h"
#include "src/error.h"

#include "src/acc/acc_ml_optimiser.h"
#include "src/acc/cuda/cuda_ml_optimiser.h"
#include "src/acc/acc_helper_functions.h"


#include "src/acc/acc_helper_functions_impl.h"
