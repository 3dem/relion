#include "hip/hip_runtime.h"
#include "src/gpu_utils/cuda_projector_plan.h"
#include "src/time.h"
#include <hip/hip_runtime.h>

//#define PP_TIMING
#ifdef PP_TIMING
    Timer timer;
	int TIMING_TOP = timer.setNew("setup");
	int TIMING_SAMPLING = 	timer.setNew(" sampling");
	int TIMING_PRIOR = 		timer.setNew("  prior");
	int TIMING_PROC_CALC = 	timer.setNew("  procCalc");
	int TIMING_PROC = 		timer.setNew("  proc");
	int TIMING_GEN = 		timer.setNew("   genOri");
	int TIMING_PERTURB = 	timer.setNew("   perturb");
	int TIMING_EULERS = 	timer.setNew(" eulers");
#define TIMING_TIC(id) timer.tic(id)
#define TIMING_TOC(id) timer.toc(id)
#else
#define TIMING_TIC(id)
#define TIMING_TOC(id)
#endif


void getOrientations(HealpixSampling &sampling, long int idir, long int ipsi, int oversampling_order,
		std::vector<RFLOAT > &my_rot, std::vector<RFLOAT > &my_tilt, std::vector<RFLOAT > &my_psi,
		std::vector<int> &pointer_dir_nonzeroprior, std::vector<RFLOAT> &directions_prior,
		std::vector<int> &pointer_psi_nonzeroprior, std::vector<RFLOAT> &psi_prior)
{
	my_rot.clear();
	my_tilt.clear();
	my_psi.clear();
	long int my_idir, my_ipsi;
	if (sampling.orientational_prior_mode == NOPRIOR)
	{
		my_idir = idir;
		my_ipsi = ipsi;
	}
	else
	{
		my_idir = pointer_dir_nonzeroprior[idir];
		my_ipsi = pointer_psi_nonzeroprior[ipsi];
	}

	if (oversampling_order == 0)
	{
		my_rot.push_back(sampling.rot_angles[my_idir]);
		my_tilt.push_back(sampling.tilt_angles[my_idir]);
		my_psi.push_back(sampling.psi_angles[my_ipsi]);
	}
	else if (!sampling.is_3D)
	{
		// for 2D sampling, only push back oversampled psi rotations
		sampling.pushbackOversampledPsiAngles(my_ipsi, oversampling_order, 0., 0., my_rot, my_tilt, my_psi);
	}
	else
	{
		// Set up oversampled grid for 3D sampling
		Healpix_Base HealPixOver(oversampling_order + sampling.healpix_order, NEST);
		int fact = HealPixOver.Nside()/sampling.healpix_base.Nside();
		int x, y, face;
		RFLOAT rot, tilt;
		// Get x, y and face for the original, coarse grid
		long int ipix = sampling.directions_ipix[my_idir];
		sampling.healpix_base.nest2xyf(ipix, x, y, face);
		// Loop over the oversampled Healpix pixels on the fine grid
		for (int j = fact * y; j < fact * (y+1); ++j)
		{
			for (int i = fact * x; i < fact * (x+1); ++i)
			{
				long int overpix = HealPixOver.xyf2nest(i, j, face);
								// this one always has to be double (also for SINGLE_PRECISION CALCULATIONS) for call to external library
				double zz, phi;
				HealPixOver.pix2ang_z_phi(overpix, zz, phi);
				rot = RAD2DEG(phi);
				tilt = ACOSD(zz);

				// The geometrical considerations about the symmetry below require that rot = [-180,180] and tilt [0,180]
				sampling.checkDirection(rot, tilt);

				sampling.pushbackOversampledPsiAngles(my_ipsi, oversampling_order, rot, tilt, my_rot, my_tilt, my_psi);
			}
		}
	}
}

template<bool invert>
__global__ void cuda_kernel_make_eulers_2D(
		XFLOAT *alphas,
		XFLOAT *eulers,
		unsigned orientation_num)
{
	unsigned oid = blockIdx.x * BLOCK_SIZE + threadIdx.x; //Orientation id

	if (oid >= orientation_num)
		return;

	XFLOAT ca, sa;
	XFLOAT a = alphas[oid] * (XFLOAT)PI / (XFLOAT)180.0;

#ifdef CUDA_DOUBLE_PRECISION
	sincos(a, &sa, &ca);
#else
	sincosf(a, &sa, &ca);
#endif

	if(!invert)
	{
		eulers[9 * oid + 0] = ca;//00
		eulers[9 * oid + 1] = sa;//01
		eulers[9 * oid + 2] = 0 ;//02
		eulers[9 * oid + 3] =-sa;//10
		eulers[9 * oid + 4] = ca;//11
		eulers[9 * oid + 5] = 0 ;//12
		eulers[9 * oid + 6] = 0 ;//20
		eulers[9 * oid + 7] = 0 ;//21
		eulers[9 * oid + 8] = 1 ;//22
	}
	else
	{
		eulers[9 * oid + 0] = ca;//00
		eulers[9 * oid + 1] =-sa;//10
		eulers[9 * oid + 2] = 0 ;//20
		eulers[9 * oid + 3] = sa;//01
		eulers[9 * oid + 4] = ca;//11
		eulers[9 * oid + 5] = 0 ;//21
		eulers[9 * oid + 6] = 0 ;//02
		eulers[9 * oid + 7] = 0 ;//12
		eulers[9 * oid + 8] = 1 ;//22
	}
}

template<bool invert,bool perturb>
__global__ void cuda_kernel_make_eulers_3D(
		XFLOAT *alphas,
		XFLOAT *betas,
		XFLOAT *gammas,
		XFLOAT *eulers,
		unsigned orientation_num,
		XFLOAT *R)
{
	XFLOAT a(0.f),b(0.f),g(0.f), A[9],B[9];
	XFLOAT ca, sa, cb, sb, cg, sg, cc, cs, sc, ss;

	unsigned oid = blockIdx.x * BLOCK_SIZE + threadIdx.x; //Orientation id

	if (oid >= orientation_num)
		return;

	for (int i = 0; i < 9; i ++)
		B[i] = (XFLOAT) 0.f;

	a = alphas[oid] * (XFLOAT)PI / (XFLOAT)180.0;
	b = betas[oid]  * (XFLOAT)PI / (XFLOAT)180.0;
	g = gammas[oid] * (XFLOAT)PI / (XFLOAT)180.0;

#ifdef CUDA_DOUBLE_PRECISION
	sincos(a, &sa, &ca);
	sincos(b,  &sb, &cb);
	sincos(g, &sg, &cg);
#else
	sincosf(a, &sa, &ca);
	sincosf(b,  &sb, &cb);
	sincosf(g, &sg, &cg);
#endif

	cc = cb * ca;
	cs = cb * sa;
	sc = sb * ca;
	ss = sb * sa;

	A[0] = ( cg * cc - sg * sa);//00
	A[1] = ( cg * cs + sg * ca);//01
	A[2] = (-cg * sb )         ;//02
	A[3] = (-sg * cc - cg * sa);//10
	A[4] = (-sg * cs + cg * ca);//11
	A[5] = ( sg * sb )         ;//12
	A[6] = ( sc )              ;//20
	A[7] = ( ss )              ;//21
	A[8] = ( cb )              ;//22


	if (perturb)
		for (int i = 0; i < 3; i++)
			for (int j = 0; j < 3; j++)
				for (int k = 0; k < 3; k++)
					B[i * 3 + j] += A[i * 3 + k] * R[k * 3 + j];
	else
		for (int i = 0; i < 9; i++)
			B[i] = A[i];

	if(invert)
	{
		eulers[9 * oid + 0] = B[0];//00
		eulers[9 * oid + 1] = B[3];//01
		eulers[9 * oid + 2] = B[6];//02
		eulers[9 * oid + 3] = B[1];//10
		eulers[9 * oid + 4] = B[4];//11
		eulers[9 * oid + 5] = B[7];//12
		eulers[9 * oid + 6] = B[2];//20
		eulers[9 * oid + 7] = B[5];//21
		eulers[9 * oid + 8] = B[8];//22
	}
	else
	{
		eulers[9 * oid + 0] = B[0];//00
		eulers[9 * oid + 1] = B[1];//10
		eulers[9 * oid + 2] = B[2];//20
		eulers[9 * oid + 3] = B[3];//01
		eulers[9 * oid + 4] = B[4];//11
		eulers[9 * oid + 5] = B[5];//21
		eulers[9 * oid + 6] = B[6];//02
		eulers[9 * oid + 7] = B[7];//12
		eulers[9 * oid + 8] = B[8];//22
	}
}

void CudaProjectorPlan::setup(
		HealpixSampling &sampling,
		std::vector<RFLOAT> &directions_prior,
		std::vector<RFLOAT> &psi_prior,
		std::vector<int> &pointer_dir_nonzeroprior,
		std::vector<int> &pointer_psi_nonzeroprior,
		MultidimArray<bool> *Mcoarse_significant,
		std::vector<RFLOAT > &pdf_class,
		std::vector<MultidimArray<RFLOAT> > &pdf_direction,
		unsigned long nr_dir,
		unsigned long nr_psi,
		unsigned long idir_min,
		unsigned long idir_max,
		unsigned long ipsi_min,
		unsigned long ipsi_max,
		unsigned long itrans_min,
		unsigned long itrans_max,
		unsigned long current_oversampling,
		unsigned long nr_oversampled_rot,
		unsigned iclass,
		bool coarse,
		bool inverseMatrix,
		bool do_skip_align,
		bool do_skip_rotate,
		int orientational_prior_mode)
{
	TIMING_TIC(TIMING_TOP);

	std::vector< RFLOAT > oversampled_rot, oversampled_tilt, oversampled_psi;

	CudaGlobalPtr<XFLOAT> alphas(nr_dir * nr_psi * nr_oversampled_rot * 9, eulers.getAllocator());
	CudaGlobalPtr<XFLOAT> betas (nr_dir * nr_psi * nr_oversampled_rot * 9, eulers.getAllocator());
	CudaGlobalPtr<XFLOAT> gammas(nr_dir * nr_psi * nr_oversampled_rot * 9, eulers.getAllocator());

	CudaGlobalPtr<XFLOAT> perturb(9, eulers.getAllocator());

	eulers.free_if_set();
	eulers.setSize(nr_dir * nr_psi * nr_oversampled_rot * 9);
	eulers.host_alloc();

	iorientclasses.free_if_set();
	iorientclasses.setSize(nr_dir * nr_psi * nr_oversampled_rot);
	iorientclasses.host_alloc();

	orientation_num = 0;

	Matrix2D<RFLOAT> R(3,3);
	RFLOAT myperturb(0.);

	if (ABS(sampling.random_perturbation) > 0.)
	{
		myperturb = sampling.random_perturbation * sampling.getAngularSampling();
		if (sampling.is_3D)
		{
			Euler_angles2matrix(myperturb, myperturb, myperturb, R);
			for (int i = 0; i < 9; i ++)
				perturb[i] = (XFLOAT) R.mdata[i];
			perturb.put_on_device();
		}
	}

	TIMING_TIC(TIMING_SAMPLING);

	for (long int idir = idir_min, iorient = 0; idir <= idir_max; idir++)
	{
		for (long int ipsi = ipsi_min, ipart = 0; ipsi <= ipsi_max; ipsi++, iorient++)
		{
			long int iorientclass = iclass * nr_dir * nr_psi + iorient;

			TIMING_TIC(TIMING_PRIOR);
			// Get prior for this direction and skip calculation if prior==0
			RFLOAT pdf_orientation;
			if (do_skip_align || do_skip_rotate)
			{
				pdf_orientation = pdf_class[iclass];
			}
			else if (orientational_prior_mode == NOPRIOR)
			{
				pdf_orientation = DIRECT_MULTIDIM_ELEM(pdf_direction[iclass], idir);
			}
			else
			{
				pdf_orientation = directions_prior[idir] * psi_prior[ipsi];
			}
			TIMING_TOC(TIMING_PRIOR);

			// In the first pass, always proceed
			// In the second pass, check whether one of the translations for this orientation of any of the particles had a significant weight in the first pass
			// if so, proceed with projecting the reference in that direction

			bool do_proceed(false);

			TIMING_TIC(TIMING_PROC_CALC);
			if (coarse && pdf_orientation > 0.)
				do_proceed = true;
			else if (pdf_orientation > 0.)
			{
				long int nr_trans = itrans_max - itrans_min + 1;
				for (long int ipart = 0; ipart < YSIZE(*Mcoarse_significant); ipart++)
				{
					long int ihidden = iorient * nr_trans;
					for (long int itrans = itrans_min; itrans <= itrans_max; itrans++, ihidden++)
					{
						if (DIRECT_A2D_ELEM(*Mcoarse_significant, ipart, ihidden))
						{
							do_proceed = true;
							break;
						}
					}
				}
			}
			TIMING_TOC(TIMING_PROC_CALC);

			TIMING_TIC(TIMING_PROC);
			if (do_proceed)
			{
				// Now get the oversampled (rot, tilt, psi) triplets
				// This will be only the original (rot,tilt,psi) triplet in the first pass (sp.current_oversampling==0)
				TIMING_TIC(TIMING_GEN);
				getOrientations(sampling, idir, ipsi, current_oversampling, oversampled_rot, oversampled_tilt, oversampled_psi,
						pointer_dir_nonzeroprior, directions_prior, pointer_psi_nonzeroprior, psi_prior);
				TIMING_TOC(TIMING_GEN);

				// Loop over all oversampled orientations (only a single one in the first pass)
				for (long int iover_rot = 0; iover_rot < nr_oversampled_rot; iover_rot++, ipart++)
				{
					if (sampling.is_3D)
					{
						alphas[orientation_num] = oversampled_rot[iover_rot];
					    betas[orientation_num]  = oversampled_tilt[iover_rot];
					    gammas[orientation_num] = oversampled_psi[iover_rot];
					}
					else
					{
						alphas[orientation_num] = oversampled_psi[iover_rot] + myperturb;
					}

					iorientclasses[orientation_num] = iorientclass;
					orientation_num ++;
				}
			}
			TIMING_TOC(TIMING_PROC);
		}
	}
	TIMING_TOC(TIMING_SAMPLING);

	iorientclasses.setSize(orientation_num);
	iorientclasses.put_on_device();

	eulers.setSize(orientation_num * 9);
	eulers.device_alloc();

	alphas.setSize(orientation_num);
	alphas.put_on_device();

	if(sampling.is_3D)
	{
		betas.setSize(orientation_num);
		betas.put_on_device();
		gammas.setSize(orientation_num);
		gammas.put_on_device();
	}

	int grid_size = ceil((float)orientation_num/(float)BLOCK_SIZE);

	if(inverseMatrix)
		if(sampling.is_3D)
			if (ABS(sampling.random_perturbation) > 0.)
				cuda_kernel_make_eulers_3D<true,true><<<grid_size,BLOCK_SIZE,0,eulers.getStream()>>>(
						~alphas,
						~betas,
						~gammas,
						~eulers,
						orientation_num,
						~perturb);
			else
				cuda_kernel_make_eulers_3D<true,false><<<grid_size,BLOCK_SIZE,0,eulers.getStream()>>>(
						~alphas,
						~betas,
						~gammas,
						~eulers,
						orientation_num,
						NULL);
		else
			cuda_kernel_make_eulers_2D<true><<<grid_size,BLOCK_SIZE,0,eulers.getStream()>>>(
					~alphas,
					~eulers,
					orientation_num);
	else
		if(sampling.is_3D)
			if (ABS(sampling.random_perturbation) > 0.)
				cuda_kernel_make_eulers_3D<false,true><<<grid_size,BLOCK_SIZE,0,eulers.getStream()>>>(
						~alphas,
						~betas,
						~gammas,
						~eulers,
						orientation_num,
						~perturb);
			else
				cuda_kernel_make_eulers_3D<false,false><<<grid_size,BLOCK_SIZE,0,eulers.getStream()>>>(
						~alphas,
						~betas,
						~gammas,
						~eulers,
						orientation_num,
						NULL);
		else
			cuda_kernel_make_eulers_2D<false><<<grid_size,BLOCK_SIZE,0,eulers.getStream()>>>(
					~alphas,
					~eulers,
					orientation_num);

	TIMING_TOC(TIMING_TOP);
}

void CudaProjectorPlan::printTo(std::ostream &os) // print
{
	os << "orientation_num = " << orientation_num << std::endl;
	os << "iorientclasses.size = " << iorientclasses.getSize() << std::endl;
	os << std::endl << "iorientclasses\tiover_rots\teulers" << std::endl;

	for (int i = 0; i < iorientclasses.getSize(); i ++)
	{
		os << iorientclasses[i] << "\t\t" << "\t";
		for (int j = 0; j < 9; j++)
			os << eulers[i * 9 + j] << "\t";
		os << std::endl;
	}
}

void CudaProjectorPlan::clear()
{
	orientation_num = 0;
	iorientclasses.free_if_set();
	iorientclasses.setSize(0);
	eulers.free_if_set();
	eulers.setSize(0);
#ifdef PP_TIMING
	timer.printTimes(false);
#endif
}
