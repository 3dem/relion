#include <signal.h>
#include <hip/hip_runtime.h>
#include "src/gpu_utils/cuda_settings.h"
#include "src/gpu_utils/cuda_backprojector.h"
#include "src/gpu_utils/cuda_device_utils.cuh"
#include "src/gpu_utils/cuda_projector.cuh"

size_t CudaBackprojector::setMdlDim(
			int xdim, int ydim, int zdim,
			int inity, int initz,
			int max_r, int paddingFactor)
{
	if (xdim != mdlX ||
		ydim != mdlY ||
		zdim != mdlZ ||
		inity != mdlInitY ||
		initz != mdlInitZ ||
		max_r != maxR ||
		paddingFactor != padding_factor)
	{
		clear();

		mdlX = xdim;
		mdlY = ydim;
		mdlZ = zdim;
		if (mdlZ < 1) mdlZ = 1;
		mdlXYZ = xdim*ydim*zdim;
		mdlInitY = inity;
		mdlInitZ = initz;
		maxR = max_r;
		maxR2 = max_r*max_r;
		padding_factor = paddingFactor;

		//Allocate space for model
		HANDLE_ERROR(hipMalloc( (void**) &d_mdlReal,   mdlXYZ * sizeof(XFLOAT)));
		HANDLE_ERROR(hipMalloc( (void**) &d_mdlImag,   mdlXYZ * sizeof(XFLOAT)));
		HANDLE_ERROR(hipMalloc( (void**) &d_mdlWeight, mdlXYZ * sizeof(XFLOAT)));

		allocaton_size = mdlXYZ * sizeof(XFLOAT) * 3;
	}

	return allocaton_size;
}

void CudaBackprojector::initMdl()
{
#ifdef CUDA_DEBUG
	if (mdlXYZ == 0)
	{
        printf("Model dimensions must be set with setMdlDim before call to setupMdl.");
        CRITICAL(ERR_MDLDIM);
	}
	if (voxelCount != 0)
	{
        printf("DEBUG_ERROR: Duplicated call to model setup");
        CRITICAL(ERR_MDLSET);
	}
#endif

	//Initiate model with zeros
	DEBUG_HANDLE_ERROR(hipMemset( d_mdlReal,   0, mdlXYZ * sizeof(XFLOAT)));
	DEBUG_HANDLE_ERROR(hipMemset( d_mdlImag,   0, mdlXYZ * sizeof(XFLOAT)));
	DEBUG_HANDLE_ERROR(hipMemset( d_mdlWeight, 0, mdlXYZ * sizeof(XFLOAT)));
}


void CudaBackprojector::getMdlData(XFLOAT *r, XFLOAT *i, XFLOAT * w)
{
	DEBUG_HANDLE_ERROR(hipStreamSynchronize(stream)); //Make sure to wait for remaining kernel executions

	DEBUG_HANDLE_ERROR(hipMemcpyAsync( r, d_mdlReal,   mdlXYZ * sizeof(XFLOAT), hipMemcpyDeviceToHost, stream));
	DEBUG_HANDLE_ERROR(hipMemcpyAsync( i, d_mdlImag,   mdlXYZ * sizeof(XFLOAT), hipMemcpyDeviceToHost, stream));
	DEBUG_HANDLE_ERROR(hipMemcpyAsync( w, d_mdlWeight, mdlXYZ * sizeof(XFLOAT), hipMemcpyDeviceToHost, stream));

	DEBUG_HANDLE_ERROR(hipStreamSynchronize(stream)); //Wait for copy
}

void CudaBackprojector::clear()
{
	mdlX = 0;
	mdlY = 0;
	mdlZ = 0;
	mdlXYZ = 0;
	mdlInitY = 0;
	mdlInitZ = 0;
	maxR = 0;
	maxR2 = 0;
	padding_factor = 0;
	allocaton_size = 0;

	if (d_mdlReal != NULL)
	{
		DEBUG_HANDLE_ERROR(hipFree(d_mdlReal));
		DEBUG_HANDLE_ERROR(hipFree(d_mdlImag));
		DEBUG_HANDLE_ERROR(hipFree(d_mdlWeight));

		d_mdlReal = d_mdlImag = d_mdlWeight = NULL;
	}
}

CudaBackprojector::~CudaBackprojector()
{
	clear();
}
